#include "hip/hip_runtime.h"
#define COLOR_RED "\x1b[1;31m"
#define COLOR_GREEN "\x1b[1;32m"
#define COLOR_RESET "\x1b[0m"

#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <vector>
#include <time.h>
#include <cmath>
#include <stdexcept>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include <chrono>

// for file writing
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <unistd.h>

// my classes
#include "Group.cpp"

#ifdef __JETBRAINS_IDE__
	#define __host__
	#define __global__
#endif

using namespace std;

/**
 6 cm/s = 125 [ms] has 30 slices
15 cm/s = 50 [ms] has 15 slices
21 cm/s = 25 [ms] has 6 slices

References:
  [1] https://en.wikipedia.org/wiki/Hodgkin-Huxley_model

**/

// parameters for variability of the simulation
const int skin_stim_time = 25;       // [ms] time of stimulating sensory (based on speed)
const int EES_FREQ = 40;             // [hz] spike frequency of EES
const int LEG_STEPS = 3;             // [step] number of full cycle steps
const float T_SIMULATION = 11 * skin_stim_time * LEG_STEPS;      // [ms] simulation time
const float INH_COEF = 1.0;          // strength coefficient of inhibitory synapses
const float SIM_STEP = 0.025;        // [s] simulation step
const bool is_4pedal = false;      // True if it is a quadrupedal simulation
const bool has_5HT = false;        // True if nuclei has 5-HT receptors

// stuff variables
unsigned int global_id = 0;          // iter to count neurons one by one
const int syn_outdegree = 27;        // synapse number outgoing from one neuron
const int neurons_in_ip = 196;       // number of neurons in interneuronal pool
const int neurons_in_moto = 169;     // motoneurons number
const int neurons_in_group = 20;     // number of neurons in a group
const int neurons_in_afferent = 120; // number of neurons in afferent

// neuron parameters
const float g_Na = 20000.0;          // [nS] Maximal conductance of the Sodium current
const float g_K = 6000.0;            // [nS] Maximal conductance of the Potassium current
const float g_L = 30.0;              // [nS] Conductance of the leak current
const float C_m = 200.0;             // [pF] Capacity of the membrane
const float E_Na = 50.0;             // [mV] Reversal potential for the Sodium current
const float E_K = -100.0;            // [mV] Reversal potential for the Potassium current
const float E_L = -72.0;             // [mV] Reversal potential for the leak current
const float E_ex = 0.0;              // [mV] Reversal potential for excitatory input
const float E_in = -80.0;            // [mV] Reversal potential for inhibitory input
const float tau_syn_exc = 0.2;       // [ms] Decay time of excitatory synaptic current (ms)
const float tau_syn_inh = 2.0;       // [ms] Decay time of inhibitory synaptic current (ms)
const float V_adj = -63.0;           // adjusts threshold to around -50 mV
const float g_bar = 1500;            // [nS] the maximal possible conductivity

unsigned int synapses_number = 0;

// calculate spike frequency in steps [steps]
const unsigned int ees_spike_each_step = (unsigned int)(1000 / EES_FREQ / SIM_STEP);
// calculate steps activation of C0 and C1
const unsigned int steps_activation_C0 = (unsigned int)(5 * skin_stim_time / SIM_STEP);
const unsigned int steps_activation_C1 = (unsigned int)(6 * skin_stim_time / SIM_STEP);
// calculate how much steps in simulation time [steps]
const unsigned int sim_time_in_steps = (unsigned int)(T_SIMULATION / SIM_STEP);

// struct for human-readable initialization of connectomes
struct SynapseMetadata {
	unsigned int post_id;           // post neuron ID
	unsigned int synapse_delay;     // [step] synaptic delay of the synapse (axonal delay is included to this delay)
	float synapse_weight;  // [nS] synaptic weight. Interpreted as changing conductivity of neuron membrane

	SynapseMetadata() = default;
	SynapseMetadata(int post_id, float synapse_delay, float synapse_weight){
		this->post_id = post_id;
		this->synapse_delay = static_cast<int>(synapse_delay * (1 / SIM_STEP) + 0.5);  // round
		this->synapse_weight = synapse_weight;
	}
};

// struct for human-readable initialization of connectomes
struct Metadata {
	Group group;
	float* voltage_array = new float[sim_time_in_steps];  // [mV] array of membrane potential
	float* g_exc = new float[sim_time_in_steps];          // [nS] array of excitatory conductivity
	float* g_inh = new float[sim_time_in_steps];          // [nS] array of inhibition conductivity
	vector<float> spike_vector;                           // [ms] spike times

	Metadata() = default;
	Metadata(Group group){
		this->group = group;
	}
};

// keep all pointers to the groups to compute their mean voltage, g_exc and etc.
vector<Metadata> all_groups;

// form structs of neurons global ID and groups name
Group form_group(string group_name, int nrns_in_group = neurons_in_group) {
	Group group = Group();
	group.group_name = group_name;     // name of a neurons group
	group.id_start = global_id;        // first ID in the group
	group.id_end = global_id + nrns_in_group - 1;  // the latest ID in the group
	group.group_size = nrns_in_group;  // size of the neurons group

	global_id += nrns_in_group;

	printf("Formed %s IDs [%d ... %d] = %d\n", group_name.c_str(), global_id - nrns_in_group, global_id - 1, nrns_in_group);

	all_groups.push_back(Metadata(group));
	return group;
}

Group EES = form_group("EES");
Group E1 = form_group("E1");
Group E2 = form_group("E2");
Group E3 = form_group("E3");
Group E4 = form_group("E4");
Group E5 = form_group("E5");

Group CV1 = form_group("CV1", 1);
Group CV2 = form_group("CV2", 1);
Group CV3 = form_group("CV3", 1);
Group CV4 = form_group("CV4", 1);
Group CV5 = form_group("CV5", 1);
Group CD4 = form_group("CD4", 1);
Group CD5 = form_group("CD5", 1);

Group OM1_0 = form_group("OM1_0");
Group OM1_1 = form_group("OM1_1");
Group OM1_2_E = form_group("OM1_2_E");
Group OM1_2_F = form_group("OM1_2_F");
Group OM1_3 = form_group("OM1_3");

Group OM2_0 = form_group("OM2_0");
Group OM2_1 = form_group("OM2_1");
Group OM2_2_E = form_group("OM2_2_E");
Group OM2_2_F = form_group("OM2_2_F");
Group OM2_3 = form_group("OM2_3");

Group OM3_0 = form_group("OM3_0");
Group OM3_1 = form_group("OM3_1");
Group OM3_2_E = form_group("OM3_2_E");
Group OM3_2_F = form_group("OM3_2_F");
Group OM3_3 = form_group("OM3_3");

Group OM4_0 = form_group("OM4_0");
Group OM4_1 = form_group("OM4_1");
Group OM4_2_E = form_group("OM4_2_E");
Group OM4_2_F = form_group("OM4_2_F");
Group OM4_3 = form_group("OM4_3");

Group OM5_0 = form_group("OM5_0");
Group OM5_1 = form_group("OM5_1");
Group OM5_2_E = form_group("OM5_2_E");
Group OM5_2_F = form_group("OM5_2_F");
Group OM5_3 = form_group("OM5_3");

Group MN_E = form_group("MN_E", neurons_in_moto);
Group MN_F = form_group("MN_F", neurons_in_moto);

Group Ia_E_aff = form_group("Ia_E_aff", neurons_in_afferent);
Group Ia_F_aff = form_group("Ia_F_aff", neurons_in_afferent);

Group R_E = form_group("R_E");
Group R_F = form_group("R_F");

Group Ia_E_pool = form_group("Ia_E_pool");
Group Ia_F_pool = form_group("Ia_F_pool");

Group eIP_E = form_group("eIP_E");
Group eIP_F = form_group("eIP_F");

Group iIP_E = form_group("iIP_E");
Group iIP_F = form_group("iIP_F");

// global vectors of SynapseMetadata of synapses for each neuron
vector<vector<SynapseMetadata>> metadatas(global_id, vector<SynapseMetadata>());

__host__
int ms_to_step(float ms) { return (int)(ms / SIM_STEP); }

__host__
float step_to_ms(int step) { return step * SIM_STEP; }

__global__
void neurons_kernel(float *V_m,
                    float *h,
                    float *m,
                    float *n,
                    float *g_exc,
                    float *g_inh,
                    bool *has_spike,
                    int *nrn_ref_time,
                    int *nrn_ref_time_timer,
                    int neurons_number,
                    int shifted_sim_iter,
                    int activated_C_,
                    int sim_iter,
                    int *begin_C_spiking,
                    int *end_C_spiking,
                    int decrease_lvl_Ia_spikes){
	// get ID of the thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Each thread gets same seed, a different sequence number, no offset
	hiprandState localState;
	hiprand_init(sim_iter, tid, 0, &localState);

	// Ia extensor/flexor IDs [1098 ... 1337], control spike number of Ia afferent by resetting neuron current
	if (965 <= tid && tid <= 1204) {
		// rule for the 2nd level
		if (decrease_lvl_Ia_spikes == 1 && tid % 3 == 0) {
			// reset current of 1/3 of neurons
			g_exc[tid] = 0;  // set maximal inhibitory conductivity
		} else {
			// rule for the 3rd level
			if (decrease_lvl_Ia_spikes == 2 && tid % 2 == 0) {
				// reset current of 1/2 of neurons
				g_exc[tid] = 0;  // set maximal inhibitory conductivity
			}
		}
	}

	// generating spikes for EES
	if (0 <= tid && tid <= 19 && (sim_iter % ees_spike_each_step == 0)) {
		g_exc[tid] = g_bar;  // set spike state
	}

	__syncthreads();

	// ignore threads which ID is greater than neurons number
	if (tid < neurons_number) {
		// reset spike flag of the current neuron before calculations
		has_spike[tid] = false;

		// Skin stimulations
		if (activated_C_ == 1) {
			// CV1
			if (tid == 120 && shifted_sim_iter > begin_C_spiking[0] && shifted_sim_iter < end_C_spiking[0] && hiprand_uniform(&localState) >= 0.5) {
				has_spike[tid] = true;
			}
			// CV2
			if (tid == 121 && shifted_sim_iter > begin_C_spiking[1] && shifted_sim_iter < end_C_spiking[1] && hiprand_uniform(&localState) >= 0.5) {
				has_spike[tid] = true;
			}
			// CV3
			if (tid == 122 && shifted_sim_iter > begin_C_spiking[2] && shifted_sim_iter < end_C_spiking[2] && hiprand_uniform(&localState) >= 0.5) {
				has_spike[tid] = true;
			}
			// CV4
			if (tid == 123 && shifted_sim_iter > begin_C_spiking[3] && shifted_sim_iter < end_C_spiking[3] && hiprand_uniform(&localState) >= 0.5) {
				has_spike[tid] = true;
			}
			// CV5
			if (tid == 124 && shifted_sim_iter > begin_C_spiking[4] && shifted_sim_iter < end_C_spiking[4] && hiprand_uniform(&localState) >= 0.5) {
				has_spike[tid] = true;
			}
		}

		// the maximal value of input current
		if (g_exc[tid] > g_bar)
			g_exc[tid] = g_bar;
		if (g_inh[tid] > g_bar)
			g_inh[tid] = g_bar;

		if (V_m[tid] > 100)
			V_m[tid] = 100;
		if (V_m[tid] < -100)
			V_m[tid] = -100;

		// use temporary V variable as V_m with adjust
		const float V = V_m[tid] - V_adj;

		// transition rates between open and closed states of the potassium channels
		float alpha_n = 0.032 * (15.0 - V) / (exp((15.0 - V) / 5.0) - 1.0);
		float beta_n = 0.5 * exp((10.0 - V) / 40.0);

		// transition rates between open and closed states of the activation of sodium channels
		float alpha_m = 0.32 * (13.0 - V) / (exp((13.0 - V) / 4.0) - 1.0);
		float beta_m = 0.28 * (V - 40.0) / (exp((V - 40.0) / 5.0) - 1.0);

		// transition rates between open and closed states of the inactivation of sodium channels
		float alpha_h = 0.128 * exp((17.0 - V) / 18.0);
		float beta_h = 4.0 / (1.0 + exp((40.0 - V) / 5.0));

		// re-calculate activation variables
		n[tid] += (alpha_n - (alpha_n + beta_n) * n[tid]) * SIM_STEP;
		m[tid] += (alpha_m - (alpha_m + beta_m) * m[tid]) * SIM_STEP;
		h[tid] += (alpha_h - (alpha_h + beta_h) * h[tid]) * SIM_STEP;

		// ToDo can be removed in future
		if (n[tid] > 1) n[tid] = 1;
		if (n[tid] < 0) n[tid] = 0;

		if (m[tid] > 1) m[tid] = 1;
		if (m[tid] < 0) m[tid] = 0;

		if (h[tid] > 1) h[tid] = 1;
		if (h[tid] < 0) h[tid] = 0;

		// ionic currents
		float I_NA = g_Na * pow(m[tid], 3) * h[tid] * (V_m[tid] - E_Na);
		float I_K = g_K * pow(n[tid], 4) * (V_m[tid] - E_K);
		float I_L = g_L * (V_m[tid] - E_L);
		float I_syn_exc = g_exc[tid] * (V_m[tid] - E_ex);
		float I_syn_inh = g_inh[tid] * (V_m[tid] - E_in);
		float dV;
		// if neuron in the refractory state -- ignore synaptic inputs. Re-calculate membrane potential
		if (nrn_ref_time_timer[tid] > 0) {
			dV = -(I_L + I_K + I_NA) / C_m * SIM_STEP;
		} else {
			dV = -(I_L + I_K + I_NA + I_syn_exc + 4 * I_syn_inh) / C_m * SIM_STEP;
		}

		if (isnan(dV)){
			printf("NAAAAAN \n");
			dV = 0;
		}

		V_m[tid] += dV;

		// re-calculate conductance
		g_exc[tid] += -g_exc[tid] / tau_syn_exc * SIM_STEP;
		g_inh[tid] += -g_inh[tid] / tau_syn_inh * SIM_STEP;

		if (V_m[tid] > 100)
			V_m[tid] = 100;
		if (V_m[tid] < -100)
			V_m[tid] = -100;

		// (threshold && not in refractory period)
		if (V_m[tid] >= -50 && nrn_ref_time_timer[tid] == 0) {
			has_spike[tid] = true;  // set spike state. It will be used in the "synapses_kernel"
			nrn_ref_time_timer[tid] = nrn_ref_time[tid];  // set the refractory period
		}

		// update the refractory period timer
		if (nrn_ref_time_timer[tid] > 0)
			nrn_ref_time_timer[tid]--;
	}
}

__global__
void synapses_kernel(bool *neuron_has_spike,     // array of bools -- is neuron has spike or not
                     float *neuron_g_exc,        // array of excitatory conductivity per neuron (changable)
                     float *neuron_g_inh,        // array of inhibitory conductivity per neuron (changable)
                     int *synapses_pre_nrn_id,   // array of pre neurons ID per synapse
                     int *synapses_post_nrn_id,  // array of post neurons ID per synapse
                     int *synapses_delay,        // array of synaptic delay per synapse
                     int *synapses_delay_timer,  // array as above but changable
                     float *synapses_weight,     // array of synaptic weight per synapse
                     int synapses_number){       // number of synapses

	// get ID of the thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// ignore threads which ID is greater than neurons number
	if (tid < synapses_number) {
		// add synaptic delay if neuron has spike
		if (synapses_delay_timer[tid] == -1 && neuron_has_spike[synapses_pre_nrn_id[tid]]) {
			synapses_delay_timer[tid] = synapses_delay[tid];
		}
		// if synaptic delay is zero it means the time when synapse increase I by synaptic weight
		if (synapses_delay_timer[tid] == 0) {
			// post neuron ID = synapses_post_nrn_id[tid][syn_id], thread-safe (!)
			if (synapses_weight[tid] >= 0) {
				atomicAdd(&neuron_g_exc[synapses_post_nrn_id[tid]], synapses_weight[tid]);
			} else {
				// remove negative sign
				atomicAdd(&neuron_g_inh[synapses_post_nrn_id[tid]], -synapses_weight[tid]);
			}
			// make synapse timer a "free" for next spikes
			synapses_delay_timer[tid] = -1;
		}
		// update synapse delay timer
		if (synapses_delay_timer[tid] > 0) {
			synapses_delay_timer[tid]--;
		}
	}
}

void connect_one_to_all( Group pre_neurons, Group post_neurons, float syn_delay, float weight) {
	std::default_random_engine generator;
	generator.seed(time(NULL));
	std::normal_distribution<float> delay_distr(syn_delay, syn_delay / 5);
	std::normal_distribution<float> weight_distr(weight, weight / 10);

	for (int pre_id = pre_neurons.id_start; pre_id <= pre_neurons.id_end; pre_id++) {
		for (int post_id = post_neurons.id_start; post_id <= post_neurons.id_end; post_id++) {
			metadatas.at(pre_id).push_back(SynapseMetadata(post_id, delay_distr(generator), weight_distr(generator)));
		}
	}

	synapses_number += pre_neurons.group_size * post_neurons.group_size;

	printf("Connect generator to %s with %s (1:%d). W=%.2f, D=%.1f\n", pre_neurons.group_name.c_str(),
	       post_neurons.group_name.c_str(), post_neurons.group_size, weight, syn_delay);
}

void connect_fixed_outdegree(Group pre_neurons, Group post_neurons, float syn_delay, float weight, int outdegree= syn_outdegree) {
	// connect neurons with uniform distribution and normal distributon for syn delay and weight
	default_random_engine generator;
	generator.seed(time(NULL));
	uniform_int_distribution<int> id_distr(post_neurons.id_start, post_neurons.id_end);
	normal_distribution<float> delay_distr(syn_delay, syn_delay / 5);
	normal_distribution<float> weight_distr(weight, weight / 10);

	#ifdef DEBUG
	printf("pre group %s (%d, %d) to post %s (%d, %d)\n",
	       pre_neurons.group_name.c_str(),
	       pre_neurons.id_start,
	       pre_neurons.id_end,
	       post_neurons.group_name.c_str(),
	       post_neurons.id_start,
	       post_neurons.id_end);
	#endif

	synapses_number += pre_neurons.group_size * outdegree;

	for (int pre_id = pre_neurons.id_start; pre_id <= pre_neurons.id_end; pre_id++) {
		for (int i = 0; i < outdegree; i++) {
			int rand_post_id = id_distr(generator);
			float syn_delay_dist = delay_distr(generator);
			if (syn_delay_dist <= 0.2) {
				syn_delay_dist = 0.2;
			}
			float syn_weight_dist = weight_distr(generator);
			#ifdef DEBUG
			printf("weight %f (%f), delay %f (%f) \n", syn_weight_dist, weight, syn_delay_dist, syn_delay);
			#endif
			metadatas.at(pre_id).push_back(SynapseMetadata(rand_post_id, syn_delay_dist, syn_weight_dist));

		}
	}

	printf("Connect %s with %s (1:%d). W=%.2f, D=%.1f\n",
	       pre_neurons.group_name.c_str(), post_neurons.group_name.c_str(), outdegree, weight, syn_delay);
}

void init_connectomes() {
	float quadru_coef = is_4pedal? 0.5 : 1;
	float sero_coef = has_5HT? 1.5 : 1;

	// input from EES
	connect_fixed_outdegree(EES, E1, 2, 500);
	connect_fixed_outdegree(E1, E2, 2, 200);
	connect_fixed_outdegree(E2, E3, 2, 200);
	connect_fixed_outdegree(E3, E4, 3, 200);
	connect_fixed_outdegree(E4, E5, 3, 200);

	connect_one_to_all(CV1, iIP_E, 0.5, 50);
	connect_one_to_all(CV2, iIP_E, 0.5, 50);
	connect_one_to_all(CV3, iIP_E, 0.5, 50);
	connect_one_to_all(CV4, iIP_E, 0.5, 50);
	connect_one_to_all(CV5, iIP_E, 0.5, 50);

	/// OM 1
	// input from EES group 1
	connect_fixed_outdegree(E1, OM1_0, 3, 17); // 7
	// input from sensory
	connect_one_to_all(CV1, OM1_0, 0.5, 12); // 18
	connect_one_to_all(CV2, OM1_0, 0.5, 12); // 18
	// [inhibition]
	connect_one_to_all(CV3, OM1_3, 1, 80);
	connect_one_to_all(CV4, OM1_3, 1, 80);
	connect_one_to_all(CV5, OM1_3, 1, 80);
	// inner connectomes
	connect_fixed_outdegree(OM1_0, OM1_1, 1, 50);
	connect_fixed_outdegree(OM1_1, OM1_2_E, 1, 24); // 23
	connect_fixed_outdegree(OM1_1, OM1_2_F, 1, 24); // 23
	connect_fixed_outdegree(OM1_1, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_2_E, OM1_1, 2.5, 20); // 22
	connect_fixed_outdegree(OM1_2_F, OM1_1, 2.5, 20); // 22
	connect_fixed_outdegree(OM1_2_E, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_2_F, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_3, OM1_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM1_3, OM1_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM1_3, OM1_2_F, 1, -70 * INH_COEF);
	// output to OM2
//	connect_fixed_outdegree(OM1_2_E, OM2_2_E, 1, 50);
	connect_fixed_outdegree(OM1_2_F, OM2_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM1_2_E, eIP_E, 1, 15, neurons_in_ip);
	connect_fixed_outdegree(OM1_2_F, eIP_F, 1, 15, neurons_in_ip);

	/// OM 2
	// input from EES group 2
	connect_fixed_outdegree(E2, OM2_0, 3, 7);
	// input from sensory [CV]
	connect_one_to_all(CV2, OM2_0, 0.5, 12);
	connect_one_to_all(CV3, OM2_0, 0.5, 12);
	// [inhibition]
	connect_one_to_all(CV4, OM2_3, 1, 80);
	connect_one_to_all(CV5, OM2_3, 1, 80);
	// inner connectomes
	connect_fixed_outdegree(OM2_0, OM2_1, 1, 50);
	connect_fixed_outdegree(OM2_1, OM2_2_E, 1, 23);
	connect_fixed_outdegree(OM2_1, OM2_2_F, 1, 23);
	connect_fixed_outdegree(OM2_1, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_2_E, OM2_1, 2.5, 20);
	connect_fixed_outdegree(OM2_2_F, OM2_1, 2.5, 20);
	connect_fixed_outdegree(OM2_2_E, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_2_F, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_3, OM2_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM2_3, OM2_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM2_3, OM2_2_F, 1, -70 * INH_COEF);
	// output to OM3
//	connect_fixed_outdegree(OM2_2_E, OM3_2_E, 1, 50);
	connect_fixed_outdegree(OM2_2_F, OM3_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM2_2_E, eIP_E, 2, 15, neurons_in_ip); // 50
	connect_fixed_outdegree(OM2_2_F, eIP_F, 2, 15, neurons_in_ip);

	/// OM 3
	// input from EES group 3
	connect_fixed_outdegree(E3, OM3_0, 3, 7);
	// input from sensory [CV]
	connect_one_to_all(CV3, OM3_0, 0.5, 13);
	connect_one_to_all(CV4, OM3_0, 0.5, 13);
	// [INH]
	connect_one_to_all(CV5, OM3_3, 1, 80);
	// input from sensory [CD]
	connect_one_to_all(CD4, OM3_0, 1, 11);
	// inner connectomes
	connect_fixed_outdegree(OM3_0, OM3_1, 1, 50);
	connect_fixed_outdegree(OM3_1, OM3_2_E, 1, 23);
	connect_fixed_outdegree(OM3_1, OM3_2_F, 1, 30);
	connect_fixed_outdegree(OM3_1, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_2_E, OM3_1, 2.5, 20);
	connect_fixed_outdegree(OM3_2_F, OM3_1, 2.5, 40);
	connect_fixed_outdegree(OM3_2_E, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_2_F, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_3, OM3_1, 1, -5 * INH_COEF);
	connect_fixed_outdegree(OM3_3, OM3_2_E, 1, -10 * INH_COEF);
	connect_fixed_outdegree(OM3_3, OM3_2_F, 1, -0.1 * INH_COEF);
	// output to OM3
//	connect_fixed_outdegree(OM3_2_E, OM4_2_E, 1, 50);
	connect_fixed_outdegree(OM3_2_F, OM4_2_F, 1, 50);
	connect_fixed_outdegree(OM3_2_E, eIP_E, 3, 15, neurons_in_ip);
	connect_fixed_outdegree(OM3_2_F, eIP_F, 3, 15, neurons_in_ip);

	/// OM 4
	// input from EES group 4
	connect_fixed_outdegree(E4, OM4_0, 3, 7);
	// input from sensory [CV]
	connect_one_to_all(CV4, OM4_0, 0.5, 12);
	connect_one_to_all(CV5, OM4_0, 0.5, 12);
	// [INH]
	// input from sensory [CD]
	connect_one_to_all(CD4, OM4_0, 1, 11);
	connect_one_to_all(CD5, OM4_0, 1, 11);
	// inner connectomes
	connect_fixed_outdegree(OM4_0, OM4_1, 3, 50);
	connect_fixed_outdegree(OM4_1, OM4_2_E, 1, 23);
	connect_fixed_outdegree(OM4_1, OM4_2_F, 1, 23);
	connect_fixed_outdegree(OM4_1, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_2_E, OM4_1, 2.5, 20);
	connect_fixed_outdegree(OM4_2_F, OM4_1, 2.5, 20);
	connect_fixed_outdegree(OM4_2_E, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_2_F, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_3, OM4_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM4_3, OM4_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM4_3, OM4_2_F, 1, -70 * INH_COEF);
	// output to OM4
//	connect_fixed_outdegree(OM4_2_E, OM5_2_E, 1, 50);
	connect_fixed_outdegree(OM4_2_F, OM5_2_F, 1, 50);
	connect_fixed_outdegree(OM4_2_E, eIP_E, 3, 13, neurons_in_ip);
	connect_fixed_outdegree(OM4_2_F, eIP_F, 2, 15, neurons_in_ip);

	/// OM 5
	// input from EES group 5
	connect_fixed_outdegree(E5, OM5_0, 3, 7);
	// input from sensory [CV]
	connect_one_to_all(CV5, OM5_0, 0.5, 18);
	// input from sensory [CD]
	connect_one_to_all(CD5, OM5_0, 1, 11);
	// inner connectomes
	connect_fixed_outdegree(OM5_0, OM5_1, 1, 50);
	connect_fixed_outdegree(OM5_1, OM5_2_E, 1, 23);
	connect_fixed_outdegree(OM5_1, OM5_2_F, 1, 23);
	connect_fixed_outdegree(OM5_1, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_2_E, OM5_1, 2.5, 22);
	connect_fixed_outdegree(OM5_2_F, OM5_1, 2.5, 22);
	connect_fixed_outdegree(OM5_2_E, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_2_F, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_3, OM5_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM5_3, OM5_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM5_3, OM5_2_F, 1, -70 * INH_COEF);
	// output to IP
	connect_fixed_outdegree(OM5_2_E, eIP_E, 2, 15, neurons_in_ip); // 15
	connect_fixed_outdegree(OM5_2_F, eIP_F, 1, 15, neurons_in_ip);

	// inhibition by C=0: IP_E, Ia_Extensor
	connect_one_to_all(iIP_E, eIP_F, 0.1, -20);
//	connect_one_to_all(iIP_E, Ia_E_aff, 0.1, -g_bar);
	// inhibition by C=0: extensor clones D1, D2, G3, D4
	connect_one_to_all(iIP_E, OM1_2_F, 0.1, -20);
	connect_one_to_all(iIP_E, OM2_2_F, 0.1, -20);
	connect_one_to_all(iIP_E, OM3_2_F, 0.1, -20);
	connect_one_to_all(iIP_E, OM4_2_F, 0.1, -20);

	// inhibition by C=1: IP_F, Ia_Flexor
	connect_one_to_all(iIP_F, eIP_E, 0.1, -20);
//	connect_one_to_all(iIP_F, Ia_F_aff, 0.1, -g_bar);

	// reflex arc
	connect_fixed_outdegree(EES, Ia_E_aff, 1, 500);
	connect_fixed_outdegree(EES, Ia_F_aff, 1, 500);

	connect_fixed_outdegree(eIP_E, MN_E, 1, 10, neurons_in_moto); // was 30

	connect_fixed_outdegree(eIP_F, MN_F, 1, 10, neurons_in_moto);

	connect_fixed_outdegree(Ia_E_aff, MN_E, 1, 5, neurons_in_moto);
	connect_fixed_outdegree(Ia_F_aff, MN_F, 1, 5, neurons_in_moto);

	/*
	// EES
	connect_fixed_outdegree(EES, Ia_E_aff, 1, 500);
	connect_fixed_outdegree(EES, Ia_F_aff, 1, 500);
	connect_fixed_outdegree(EES, E1, 2, 500);
	// E1
	connect_fixed_outdegree(E1, E2, 2, 200);
	connect_fixed_outdegree(E1, OM1_0, 1, 15);
	// E2
	connect_fixed_outdegree(E2, E3, 2, 200);
	connect_fixed_outdegree(E2, OM2_0, 1, 15);
	// E3
	connect_fixed_outdegree(E3, E4, 3, 200);
	connect_fixed_outdegree(E3, OM3_0, 1, 15);
	// E4
	connect_fixed_outdegree(E4, E5, 3, 200);
	connect_fixed_outdegree(E4, OM4_0, 1, 15);
	// E5
	connect_fixed_outdegree(E5, OM5_0, 1, 15);

	// CV1
	// [excatation]
	connect_fixed_outdegree(CV1, iIP_E, 1, 5);
	connect_fixed_outdegree(CV1, OM1_0, 0.5, 3); // 5
	// CV2
	// [excatation]
	connect_fixed_outdegree(CV2, iIP_E, 1, 5);
	connect_fixed_outdegree(CV2, OM1_0, 0.5, 3); // 5
	connect_fixed_outdegree(CV2, OM2_0, 0.5, 5);
	// CV3
	// [excatation]
	connect_fixed_outdegree(CV3, iIP_E, 1, 5);
	connect_fixed_outdegree(CV3, OM2_0, 0.5, 5);
	connect_fixed_outdegree(CV3, OM3_0, 0.5, 5);
	// [inhibition]
	connect_fixed_outdegree(CV3, OM1_3, 1, 5);
	// CV4
	// [excatation]
	connect_fixed_outdegree(CV4, iIP_E, 1, 5);
	connect_fixed_outdegree(CV4, OM3_0, 0.5, 5);
	connect_fixed_outdegree(CV4, OM4_0, 0.5, 5);
	// [inhibition]
	connect_fixed_outdegree(CV4, OM1_3, 1, 5);
	connect_fixed_outdegree(CV4, OM2_3, 1, 5);
	// CV5
	// [excatation]
	connect_fixed_outdegree(CV5, iIP_E, 1, 5);
	connect_fixed_outdegree(CV5, OM4_0, 0.5, 5);
	connect_fixed_outdegree(CV5, OM5_0, 0.5, 5);
	// [inhibition]
	connect_fixed_outdegree(CV5, OM1_3, 1, 5);
	connect_fixed_outdegree(CV5, OM2_3, 1, 5);
	connect_fixed_outdegree(CV5, OM3_3, 1, 5);

	// CD4 [for 4pedal simulations]
	connect_fixed_outdegree(CD4, OM3_0, 1, 5);
	connect_fixed_outdegree(CD4, OM4_0, 1, 5);
	// CD5 [for 4pedal simulations]
	connect_fixed_outdegree(CD5, OM4_0, 1, 5);
	connect_fixed_outdegree(CD5, OM5_0, 1, 5);

	/// OM 1
	// inner connectomes
	connect_fixed_outdegree(OM1_0, OM1_1, 1, 7);
	connect_fixed_outdegree(OM1_1, OM1_2_E, 1, 24); // 23
	connect_fixed_outdegree(OM1_1, OM1_2_F, 1, 24); // 23
	connect_fixed_outdegree(OM1_1, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_2_E, OM1_1, 2.5, 23); // 22
	connect_fixed_outdegree(OM1_2_F, OM1_1, 2.5, 23); // 22
	connect_fixed_outdegree(OM1_2_E, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_2_F, OM1_3, 1, 3);
	connect_fixed_outdegree(OM1_3, OM1_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM1_3, OM1_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM1_3, OM1_2_F, 1, -70 * INH_COEF);
	// output to OM2
	connect_fixed_outdegree(OM1_2_E, OM2_2_E, 1, 50);
	connect_fixed_outdegree(OM1_2_F, OM2_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM1_2_E, eIP_E, 1, 15, neurons_in_ip);
	connect_fixed_outdegree(OM1_2_F, eIP_F, 3, 2, neurons_in_ip);

	/// OM 2
	// inner connectomes
	connect_fixed_outdegree(OM2_0, OM2_1, 1, 50);
	connect_fixed_outdegree(OM2_1, OM2_2_E, 1, 23);
	connect_fixed_outdegree(OM2_1, OM2_2_F, 1, 23);
	connect_fixed_outdegree(OM2_1, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_2_E, OM2_1, 2.5, 22);
	connect_fixed_outdegree(OM2_2_F, OM2_1, 2.5, 22);
	connect_fixed_outdegree(OM2_2_E, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_2_F, OM2_3, 1, 3);
	connect_fixed_outdegree(OM2_3, OM2_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM2_3, OM2_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM2_3, OM2_2_F, 1, -70 * INH_COEF);
	// output to OM3
	connect_fixed_outdegree(OM2_2_E, OM3_2_E, 1, 50);
	connect_fixed_outdegree(OM2_2_F, OM3_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM2_2_E, eIP_E, 2, 15, neurons_in_ip);
	connect_fixed_outdegree(OM2_2_F, eIP_F, 2, 3, neurons_in_ip);

	/// OM 3
	// inner connectomes
	connect_fixed_outdegree(OM3_0, OM3_1, 1, 50);
	connect_fixed_outdegree(OM3_1, OM3_2_E, 1, 24);
	connect_fixed_outdegree(OM3_1, OM3_2_F, 1, 30);
	connect_fixed_outdegree(OM3_1, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_2_E, OM3_1, 2.5, 22);
	connect_fixed_outdegree(OM3_2_F, OM3_1, 2.5, 40);
	connect_fixed_outdegree(OM3_2_E, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_2_F, OM3_3, 1, 3);
	connect_fixed_outdegree(OM3_3, OM3_1, 1, -5 * INH_COEF);
	connect_fixed_outdegree(OM3_3, OM3_2_E, 1, -10 * INH_COEF);
	connect_fixed_outdegree(OM3_3, OM3_2_F, 1, -0.1 * INH_COEF);
	// output to OM4
	connect_fixed_outdegree(OM3_2_E, OM4_2_E, 1, 50);
	connect_fixed_outdegree(OM3_2_F, OM4_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM3_2_E, eIP_E, 3, 15, neurons_in_ip);
	connect_fixed_outdegree(OM3_2_F, eIP_F, 3, 6, neurons_in_ip);

	/// OM 4
	// inner connectomes
	connect_fixed_outdegree(OM4_0, OM4_1, 3, 50);
	connect_fixed_outdegree(OM4_1, OM4_2_E, 1, 23);
	connect_fixed_outdegree(OM4_1, OM4_2_F, 1, 23);
	connect_fixed_outdegree(OM4_1, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_2_E, OM4_1, 2.5, 22);
	connect_fixed_outdegree(OM4_2_F, OM4_1, 2.5, 22);
	connect_fixed_outdegree(OM4_2_E, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_2_F, OM4_3, 1, 3);
	connect_fixed_outdegree(OM4_3, OM4_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM4_3, OM4_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM4_3, OM4_2_F, 1, -70 * INH_COEF);
	// output to OM5
	connect_fixed_outdegree(OM4_2_E, OM5_2_E, 1, 50);
	connect_fixed_outdegree(OM4_2_F, OM5_2_F, 1, 50);
	// output to IP
	connect_fixed_outdegree(OM4_2_E, eIP_E, 3, 13, neurons_in_ip);
	connect_fixed_outdegree(OM4_2_F, eIP_F, 1, 6, neurons_in_ip);

	/// OM 5
	// inner connectomes
	connect_fixed_outdegree(OM5_0, OM5_1, 1, 50);
	connect_fixed_outdegree(OM5_1, OM5_2_E, 1, 23);
	connect_fixed_outdegree(OM5_1, OM5_2_F, 1, 23);
	connect_fixed_outdegree(OM5_1, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_2_E, OM5_1, 2.5, 22);
	connect_fixed_outdegree(OM5_2_F, OM5_1, 2.5, 22);
	connect_fixed_outdegree(OM5_2_E, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_2_F, OM5_3, 1, 3);
	connect_fixed_outdegree(OM5_3, OM5_1, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM5_3, OM5_2_E, 1, -70 * INH_COEF);
	connect_fixed_outdegree(OM5_3, OM5_2_F, 1, -70 * INH_COEF);
	// output to IP
	connect_fixed_outdegree(OM5_2_E, eIP_E, 2, 15, neurons_in_ip);
	connect_fixed_outdegree(OM5_2_F, eIP_F, 3, 3, neurons_in_ip);

	/// Reflex arc
	// inhibition by iIP_E (C0). Aff lens implemented as hard code inhibiting: LiIP_E => Ia_aff --*--> MN_E
	connect_fixed_outdegree(iIP_E, eIP_F, 1, -10 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(iIP_E, OM1_2_F, 1, -10 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(iIP_E, OM2_2_F, 1, -10 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(iIP_E, OM3_2_E, 1, -10 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(iIP_E, OM4_2_F, 1, -10 * INH_COEF, neurons_in_ip);

	// inhibition by iIP_F (C1). Aff lens implemented as hard code inhibiting: iIP_F => Ia_aff --*--> MN_F
	connect_fixed_outdegree(iIP_F, eIP_E, 1, -10 * INH_COEF, neurons_in_ip);

	// eIP_E connectomes
	connect_fixed_outdegree(eIP_E, MN_E, 1, 2, neurons_in_ip);
//	connect_fixed_outdegree(eIP_E, iIP_E, 1, 10, neurons_in_ip);

	// eIP_F connectomes
	connect_fixed_outdegree(eIP_F, MN_E, 1, 2, neurons_in_ip);
	connect_fixed_outdegree(eIP_F, iIP_F, 1, 10, neurons_in_ip);

	// Ia_E_pool
	connect_fixed_outdegree(Ia_E_pool, Ia_F_pool, 1, -1 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(Ia_E_pool, MN_F, 1, -1 * INH_COEF, neurons_in_ip);

	// Ia_F_pool
	connect_fixed_outdegree(Ia_F_pool, Ia_E_pool, 1, -1 * INH_COEF, neurons_in_ip);
	connect_fixed_outdegree(Ia_F_pool, MN_E, 1, -1 * INH_COEF, neurons_in_ip);

	// R_E
	connect_fixed_outdegree(R_E, R_F, 1, -1 * INH_COEF);
	connect_fixed_outdegree(R_E, MN_E, 1, -1 * INH_COEF);
	connect_fixed_outdegree(R_E, Ia_E_pool, 1, -1 * INH_COEF);

	// R_F
	connect_fixed_outdegree(R_F, R_E, 1, -1 * INH_COEF);
	connect_fixed_outdegree(R_F, MN_F, 1, -1 * INH_COEF);
	connect_fixed_outdegree(R_F, Ia_F_pool, 1, -1 * INH_COEF);

	// Ia_E_aff
	connect_fixed_outdegree(Ia_E_aff, MN_E, 1, 5, neurons_in_afferent);
	connect_fixed_outdegree(Ia_E_aff, Ia_E_pool, 1, 2, neurons_in_afferent);

	// Ia_F_aff
	connect_fixed_outdegree(Ia_F_aff, MN_F, 1, 5, neurons_in_afferent);
	connect_fixed_outdegree(Ia_F_aff, Ia_F_pool, 1, 2, neurons_in_afferent);*/
}

void save(int test_index, Metadata metadata){
	ofstream file;
	string file_name = "/dat/" + to_string(test_index) + "_" + metadata.group.group_name + ".dat";
	file.open("/home/alex/GitHub/memristive-spinal-cord/GRAS/matrix_solution/" + file_name);
	// save voltage
	for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++)
		file << metadata.voltage_array[sim_iter] << " ";
	file << endl;

	// save g_exc
	for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++)
		file << metadata.g_exc[sim_iter] << " ";
	file << endl;

	// save g_inh
	for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++)
		file << metadata.g_inh[sim_iter] << " ";
	file << endl;

	// save spikes
	for (float &value: metadata.spike_vector) {
		file << value << " ";
	}
	file.close();

	printf("saved %s \n", metadata.group.group_name.c_str());
}

void save_result(int test_index, int full_save) {
	char cwd[256];

	getcwd(cwd, sizeof(cwd));
	printf("[Test #%d] Save %s results to: %s \n", test_index, (full_save == 0)? "only MOTO" : "ALL", cwd);

	for(Metadata metadata : all_groups) {
		if (full_save == 0) {
			if(metadata.group.group_name == "MN_E")
				save(test_index, metadata);
			if(metadata.group.group_name == "MN_F")
				save(test_index, metadata);
		} else {
			save(test_index, metadata);
		}
	}
}

// copy data from host to device
template <typename type>
void memcpyHtD(type* gpu, type* host, int size) {
	hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice);
}

// copy data from device to host
template <typename type>
void memcpyDtH(type* host, type* gpu, int size) {
	hipMemcpy(host, gpu, sizeof(type) * size, hipMemcpyDeviceToHost);
}

// get datasize of current variable type and its number
template <typename type>
unsigned int datasize(int size) {
	return sizeof(type) * size;
}

// fill array with current value
template <typename type>
void init_array(type *array, int size, type value) {
	for(int i = 0; i < size; i++)
		array[i] = value;
}

// fill array with normal distribution values
template <typename type>
void rand_normal_init_array(type *array, int size, type mean, type stddev) {
	default_random_engine generator;
	generator.seed(time(NULL));
	normal_distribution<float> normal_dist(mean, stddev);

	for(int i = 0; i < size; i++)
		array[i] = normal_dist(generator);
}

__host__
void simulate(int test_index, int full_save) {
	chrono::time_point<chrono::system_clock> simulation_t_start, simulation_t_end;
	const int neurons_number = static_cast<int>(metadatas.size());

	// init connectomes
	init_connectomes();

	/// CPU variables
	// neuron variables
	float nrn_n[neurons_number];             // dimensionless quantity [0 .. 1] of potassium channel activation
	float nrn_h[neurons_number];             // dimensionless quantity [0 .. 1] of sodium channel activation
	float nrn_m[neurons_number];             // dimensionless quantity [0 .. 1] of sodium channel inactivation
	float nrn_v_m[neurons_number];           // [mV] neuron membrane potential
	float nrn_g_exc[neurons_number];         // [nS] excitatory synapse exponential conductance
	float nrn_g_inh[neurons_number];         // [nS] inhibitory synapse exponential conductance
	bool nrn_has_spike[neurons_number];      // neuron state - has spike or not
	int nrn_ref_time[neurons_number];        // [step] neuron refractory time
	int nrn_ref_time_timer[neurons_number];  // [step] neuron refractory time timer

	int begin_C_spiking[5] = {ms_to_step(0),
                              ms_to_step(skin_stim_time),
                              ms_to_step(2 * skin_stim_time),
                              ms_to_step(3 * skin_stim_time),
                              ms_to_step(5 * skin_stim_time)};
	int end_C_spiking[5] = {ms_to_step(skin_stim_time - 0.1),
                            ms_to_step(2 * skin_stim_time - 0.1),
                            ms_to_step(3 * skin_stim_time - 0.1),
                            ms_to_step(5 * skin_stim_time - 0.1),
                            ms_to_step(6 * skin_stim_time - 0.1)};

	// fill arrays by initial data
	init_array<float>(nrn_n, neurons_number, 0);      // by default neurons have closed potassium channel
	init_array<float>(nrn_h, neurons_number, 1);      // by default neurons have opened sodium channel activation
	init_array<float>(nrn_m, neurons_number, 0);      // by default neurons have closed sodium channel inactivation
	init_array<float>(nrn_v_m, neurons_number, E_L);  // by default neurons have E_L membrane state at start
	init_array<float>(nrn_g_exc, neurons_number, 0);  // by default neurons have zero excitatory synaptic conductivity
	init_array<float>(nrn_g_inh, neurons_number, 0);  // by default neurons have zero inhibitory synaptic conductivity
	init_array<bool>(nrn_has_spike, neurons_number, false);         // by default neurons haven't spikes at start
	init_array<int>(nrn_ref_time, neurons_number, ms_to_step(3.0)); // by default neurons have 3ms refractory period
	init_array<int>(nrn_ref_time_timer, neurons_number, 0);         // by default neurons have ref_t timers as 0
	// rand_normal_init_array<float>(nrn_c_m, neurons_number, 200, 5)

	// synapse variables
	unsigned int syn_id = 0;
	int synapses_pre_nrn_id[synapses_number];
	int synapses_post_nrn_id[synapses_number];
	float synapses_weight[synapses_number];
	int synapses_delay[synapses_number];
	int synapses_delay_timer[synapses_number];

	// fill arrays of synapses
	for(int neuron_id = 0; neuron_id < neurons_number; neuron_id++) {
		// for each metadata of current neuron fill data in array to copy in future pointers to the GPU
		for(SynapseMetadata metadata : metadatas.at(neuron_id)) {
			synapses_pre_nrn_id[syn_id] = neuron_id;
			synapses_post_nrn_id[syn_id] = metadata.post_id;
			synapses_delay[syn_id] = metadata.synapse_delay;
			synapses_weight[syn_id] = metadata.synapse_weight;
			syn_id++;
		}
	}
	init_array<int>(synapses_delay_timer, synapses_number, -1);

	// neuron variables
	float* gpu_nrn_n;
	float* gpu_nrn_h;
	float* gpu_nrn_m;
	float* gpu_nrn_v_m;
	float* gpu_nrn_g_exc;
	float* gpu_nrn_g_inh;
	bool* gpu_nrn_has_spike;
	int* gpu_nrn_ref_time;
	int* gpu_nrn_ref_time_timer;

	// synapse variables
	int* gpu_syn_pre_nrn_id;
	int* gpu_syn_post_nrn_id;
	float* gpu_syn_weight;
	int* gpu_syn_delay;
	int* gpu_syn_delay_timer;

	int *gpu_begin_C_spiking;
	int *gpu_end_C_spiking;

	// allocate memory in the GPU
	hipMalloc(&gpu_nrn_n, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_h, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_m, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_v_m, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_g_exc, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_g_inh, datasize<float>(neurons_number));
	hipMalloc(&gpu_nrn_has_spike, datasize<bool>(neurons_number));
	hipMalloc(&gpu_nrn_ref_time, datasize<int>(neurons_number));
	hipMalloc(&gpu_nrn_ref_time_timer, datasize<int>(neurons_number));

	hipMalloc(&gpu_syn_pre_nrn_id, datasize<int>(synapses_number));
	hipMalloc(&gpu_syn_post_nrn_id, datasize<int>(synapses_number));
	hipMalloc(&gpu_syn_weight, datasize<float>(synapses_number));
	hipMalloc(&gpu_syn_delay, datasize<int>(synapses_number));
	hipMalloc(&gpu_syn_delay_timer, datasize<int>(synapses_number));

	hipMalloc(&gpu_begin_C_spiking, datasize<int>(5));
	hipMalloc(&gpu_end_C_spiking, datasize<int>(5));

	// copy data from CPU to GPU
	memcpyHtD<float>(gpu_nrn_n, nrn_n, neurons_number);
	memcpyHtD<float>(gpu_nrn_h, nrn_h, neurons_number);
	memcpyHtD<float>(gpu_nrn_m, nrn_m, neurons_number);
	memcpyHtD<float>(gpu_nrn_v_m, nrn_v_m, neurons_number);
	memcpyHtD<float>(gpu_nrn_g_exc, nrn_g_exc, neurons_number);
	memcpyHtD<float>(gpu_nrn_g_inh, nrn_g_inh, neurons_number);
	memcpyHtD<bool>(gpu_nrn_has_spike, nrn_has_spike, neurons_number);
	memcpyHtD<int>(gpu_nrn_ref_time, nrn_ref_time, neurons_number);
	memcpyHtD<int>(gpu_nrn_ref_time_timer, nrn_ref_time_timer, neurons_number);

	memcpyHtD<int>(gpu_syn_pre_nrn_id, synapses_pre_nrn_id, synapses_number);
	memcpyHtD<int>(gpu_syn_post_nrn_id, synapses_post_nrn_id, synapses_number);
	memcpyHtD<float>(gpu_syn_weight, synapses_weight, synapses_number);
	memcpyHtD<int>(gpu_syn_delay, synapses_delay, synapses_number);
	memcpyHtD<int>(gpu_syn_delay_timer, synapses_delay_timer, synapses_number);

	memcpyHtD<int>(gpu_begin_C_spiking, begin_C_spiking, 5);
	memcpyHtD<int>(gpu_end_C_spiking, end_C_spiking, 5);

	// preparations for simulation
	int threads_per_block = 512;
	int nrn_num_blocks = neurons_number / threads_per_block + 1;
	int syn_num_blocks = synapses_number / threads_per_block + 1;

	printf("Size of network: %d, Synapse number: %d\n", neurons_number, synapses_number);
	printf("Start GPU with %d threads x %d (nrn) / %d (syn) (Total: %d nrn threads / %d syn threads) \n",
	       threads_per_block, nrn_num_blocks, syn_num_blocks, threads_per_block * nrn_num_blocks, threads_per_block * syn_num_blocks);

	// stuff variables for controlling C0/C1 activation
	int activated_C_ = 0;
	int shift_time_by_step = 0;
	int local_iter = 0;
	int decrease_lvl_Ia_spikes;

	simulation_t_start = chrono::system_clock::now();

	// the main simulation loop
	for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++) {
		decrease_lvl_Ia_spikes = 0;

		// if flexor C0 activated, find the end of it and change to C1
		if (activated_C_ == 0) {
			if (local_iter != 0 && local_iter % steps_activation_C0 == 0) {
				activated_C_ = 1; // change to C1
				local_iter = 0;   // reset local time iterator
				shift_time_by_step += steps_activation_C0;  // add constant 125 ms
			}
		// if extensor C1 activated, find the end of it and change to C0
		} else {
			if (local_iter != 0 && local_iter % steps_activation_C1 == 0) {
				activated_C_ = 0; // change to C0
				local_iter = 0;   // reset local time iterator
				shift_time_by_step += steps_activation_C1;  // add time equal to n_layers * 25 ms
			}
		}

		int shifted_iter_time = sim_iter - shift_time_by_step;

		// CV1
		if ((begin_C_spiking[0] <= shifted_iter_time) && (shifted_iter_time < end_C_spiking[0])) {
			decrease_lvl_Ia_spikes = 2;
		} else {
			// CV2
			if ((begin_C_spiking[1] <= shifted_iter_time) && (shifted_iter_time < end_C_spiking[1])) {
				decrease_lvl_Ia_spikes = 1;
			} else {
				// CV3
				if ((begin_C_spiking[2] <= shifted_iter_time) && (shifted_iter_time < end_C_spiking[2])) {
					decrease_lvl_Ia_spikes = 0;
				} else {
					// CV4
					if ((begin_C_spiking[3] <= shifted_iter_time) && (shifted_iter_time < end_C_spiking[3])) {
						decrease_lvl_Ia_spikes = 1;
					} else {
						// CV5
						if ((begin_C_spiking[4] <= shifted_iter_time) && (shifted_iter_time < end_C_spiking[4])) {
							decrease_lvl_Ia_spikes = 2;
						}
					}
				}
			}
		}

		// update local iter (warning: can be resetted at C0/C1 activation)
		local_iter++;

		// invoke GPU kernel for neurons
		neurons_kernel<<<nrn_num_blocks, threads_per_block>>>(gpu_nrn_v_m,
		                                                      gpu_nrn_h,
		                                                      gpu_nrn_m,
		                                                      gpu_nrn_n,
		                                                      gpu_nrn_g_exc,
		                                                      gpu_nrn_g_inh,
		                                                      gpu_nrn_has_spike,
		                                                      gpu_nrn_ref_time,
		                                                      gpu_nrn_ref_time_timer,
		                                                      neurons_number,
		                                                      sim_iter - shift_time_by_step,
		                                                      activated_C_,
		                                                      sim_iter,
		                                                      gpu_begin_C_spiking,
		                                                      gpu_end_C_spiking,
		                                                      decrease_lvl_Ia_spikes);

		// copy data from GPU
		memcpyDtH<float>(nrn_v_m, gpu_nrn_v_m, neurons_number);
		memcpyDtH<float>(nrn_g_exc, gpu_nrn_g_exc, neurons_number);
		memcpyDtH<float>(nrn_g_inh, gpu_nrn_g_inh, neurons_number);
		memcpyDtH<bool>(nrn_has_spike, gpu_nrn_has_spike, neurons_number);

		// fill records arrays
		for(Metadata &metadata : all_groups) {
			float nrn_mean_volt = 0;
			float nrn_mean_g_exc = 0;
			float nrn_mean_g_inh = 0;

			for(int tid = metadata.group.id_start; tid <= metadata.group.id_end; tid++) {
				nrn_mean_volt += nrn_v_m[tid];
				nrn_mean_g_exc += nrn_g_exc[tid];
				nrn_mean_g_inh += nrn_g_inh[tid];
				if (nrn_has_spike[tid]) {
					metadata.spike_vector.push_back(step_to_ms(sim_iter) + 0.25);
				}
			}
			metadata.voltage_array[sim_iter] = nrn_mean_volt / metadata.group.group_size;
			metadata.g_exc[sim_iter] = nrn_mean_g_exc / metadata.group.group_size;
			metadata.g_inh[sim_iter] = nrn_mean_g_inh / metadata.group.group_size;
		}

		// invoke GPU kernel for synapses
		synapses_kernel<<<syn_num_blocks, threads_per_block>>>(gpu_nrn_has_spike,
		                                                       gpu_nrn_g_exc,
		                                                       gpu_nrn_g_inh,
		                                                       gpu_syn_pre_nrn_id,
		                                                       gpu_syn_post_nrn_id,
		                                                       gpu_syn_delay,
		                                                       gpu_syn_delay_timer,
		                                                       gpu_syn_weight,
		                                                       synapses_number);
	} // end of the simulation iteration loop

	simulation_t_end = chrono::system_clock::now();

	hipDeviceSynchronize();  // tell the CPU to halt further processing until the CUDA has finished doing its business
	hipDeviceReset();  // remove all all device allocations (destroy a CUDA context)

	// save recorded data
	save_result(test_index, full_save);

	auto sim_time_diff = chrono::duration_cast<chrono::milliseconds>(simulation_t_end - simulation_t_start).count();
	printf("Elapsed %li ms (measured) | T_sim = %.2f ms\n", sim_time_diff, T_SIMULATION);
	printf("%s x%f\n", (double)(T_SIMULATION / sim_time_diff) > 1?
	                   COLOR_GREEN "faster" COLOR_RESET: COLOR_RED "slower" COLOR_RESET, T_SIMULATION / sim_time_diff);
}

// runner
int main(int argc, char* argv[]) {
	// arg 1: test index (0 equal to test with i=0, 1 equal to tests with i=0,1)
	// arg 2: 0 - save only motoneurons V_m, 1 - save all results
	simulate(atoi(argv[1]), atoi(argv[2]));

	return 0;
}