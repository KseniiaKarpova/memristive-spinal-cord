#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <vector>
#include <ctime>
#include <stdexcept>
#include <random>

#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <unistd.h>

#include <thread>
#include <chrono>

#include "Group.cpp"

#ifdef __JETBRAINS_IDE__
	#define __host__
	#define __global__
#endif

using namespace std;

const unsigned int syn_outdegree = 27;
const unsigned int neurons_in_ip = 196;
const unsigned int neurons_in_moto = 169;
const unsigned int neurons_in_group = 20;
const unsigned int neurons_in_afferent = 196;

const int skin_stim_time = 25;
const float INH_COEF = 1.0f;

// 6 CMS = 125 [ms]
// 15 CMS = 50 [ms]
// 21 CMS = 25 [ms]

// stuff variable
unsigned int global_id = 0;
const float T_sim = 1000;
const float SIM_STEP = 0.25;
const unsigned int sim_time_in_step = (unsigned int)(T_sim / SIM_STEP);

__host__
int ms_to_step(float ms) { return (int)(ms / SIM_STEP); }

struct SynapseMetadata{
	// struct for human-readable initialization of connectomes
	int post_id;
	int synapse_delay;
	float synapse_weight;

	SynapseMetadata() = default;
	SynapseMetadata(int post_id, float synapse_delay, float synapse_weight){
		this->post_id = post_id;
		this->synapse_delay = static_cast<int>(synapse_delay * (1 / SIM_STEP) + 0.5); // round
		this->synapse_weight = synapse_weight;
	}
};

Group form_group(string group_name, int nrns_in_group = neurons_in_group) {
	// form structs of neurons global ID and groups name
	Group group = Group();

	group.group_name = group_name;
	group.id_start = global_id;
	group.id_end = global_id + nrns_in_group - 1;
	group.group_size = nrns_in_group;

	global_id += nrns_in_group;

	printf("Formed %s IDs [%d ... %d] = %d\n",
		   group_name.c_str(), global_id - nrns_in_group, global_id - 1, nrns_in_group);

	return group;
}

// Form neuron groups
Group C1 = form_group("C1");
Group C2 = form_group("C2");
Group C3 = form_group("C3");
Group C4 = form_group("C4");
Group C5 = form_group("C5");

Group D1_1 = form_group("D1_1");
Group D1_2 = form_group("D1_2");
Group D1_3 = form_group("D1_3");
Group D1_4 = form_group("D1_4");

Group D2_1 = form_group("D2_1");
Group D2_2 = form_group("D2_2");
Group D2_3 = form_group("D2_3");
Group D2_4 = form_group("D2_4");

Group D3_1 = form_group("D3_1");
Group D3_2 = form_group("D3_2");
Group D3_3 = form_group("D3_3");
Group D3_4 = form_group("D3_4");

Group D4_1 = form_group("D4_1");
Group D4_2 = form_group("D4_2");
Group D4_3 = form_group("D4_3");
Group D4_4 = form_group("D4_4");

Group D5_1 = form_group("D5_1");
Group D5_2 = form_group("D5_2");
Group D5_3 = form_group("D5_3");
Group D5_4 = form_group("D5_4");

Group G1_1 = form_group("G1_1");
Group G1_2 = form_group("G1_2");
Group G1_3 = form_group("G1_3");

Group G2_1 = form_group("G2_1");
Group G2_2 = form_group("G2_2");
Group G2_3 = form_group("G2_3");

Group G3_1 = form_group("G3_1");
Group G3_2 = form_group("G3_2");
Group G3_3 = form_group("G3_3");

Group G4_1 = form_group("G4_1");
Group G4_2 = form_group("G4_2");
Group G4_3 = form_group("G4_3");

Group G5_1 = form_group("G5_1");
Group G5_2 = form_group("G5_2");
Group G5_3 = form_group("G5_3");

Group IP_E = form_group("IP_E", neurons_in_ip);
Group MP_E = form_group("MP_E", neurons_in_moto);
Group EES = form_group("EES");
Group Ia = form_group("Ia", neurons_in_afferent);

Group inh_group3 = form_group("inh_group3");
Group inh_group4 = form_group("inh_group4");
Group inh_group5 = form_group("inh_group5");

Group ees_group1 = form_group("ees_group1");
Group ees_group2 = form_group("ees_group2");
Group ees_group3 = form_group("ees_group3");
Group ees_group4 = form_group("ees_group4");

// Global vectors of SynapseMetadata of synapses for each neuron
vector<vector<SynapseMetadata>> metadatas(global_id, vector<SynapseMetadata>());

// Parameters (const)
const float C = 100;        // [pF] membrane capacitance
const float V_rest = -72;   // [mV] resting membrane potential
const float V_thld = -55;   // [mV] spike threshold
const float k = 0.7;          // [pA * mV-1] constant ("1/R")
const float a = 0.02;         // [ms-1] time scale of the recovery variable U_m. Higher a, the quicker recovery
const float b = 0.2;          // [pA * mV-1] sensitivity of U_m to the sub-threshold fluctuations of the V_m
const float c = -80;        // [mV] after-spike reset value of V_m
const float d = 6;          // [pA] after-spike reset value of U_m
const float V_peak = 35;    // [mV] spike cutoff value

const unsigned int steps_activation_C0 = (unsigned int)(5 * skin_stim_time / SIM_STEP);
const unsigned int steps_activation_C1 = (unsigned int)(6 * skin_stim_time / SIM_STEP);

__global__
void sim_kernel(float* old_v,
				float* old_u,
				float* nrn_current,
				int* nrn_ref_time,
				int* nrn_ref_time_timer,
				int* synapses_number,
				bool* has_spike,
				int** synapses_post_nrn_id,
				int** synapses_delay,
				int** synapses_delay_timer,
				float** synapses_weight,
				unsigned int nrn_size,
				float* voltage_recording,
				int activated_C_,
				int shift_time_by_step,
				int sim_iter) {

	// get id of the thread
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

	if (activated_C_ == 0){
		return;
	}

	// neuron (tid = neuron id) stride loop (0, 1024, 1, 1025 ...)
	for (int tid = thread_id; tid < nrn_size; tid += blockDim.x * gridDim.x) {
		// C1
		if (0 <= tid && tid <= 19
			&& shift_time_by_step <= sim_iter
			&& sim_iter < (100 + shift_time_by_step)
			&& (sim_iter % 20 == 0)) {
			nrn_current[tid] = 5000; // enough for spike
		}
		// C2
		if (20 <= tid && tid <= 39
			&& (100 + shift_time_by_step) <= sim_iter
			&& sim_iter < (200 + shift_time_by_step)
			&& (sim_iter % 20 == 0)) {
			nrn_current[tid] = 5000; // enough for spike
		}
		// C3
		if (40 <= tid && tid <= 59
			&& (200 + shift_time_by_step) <= sim_iter
			&& sim_iter < (300 + shift_time_by_step)
			&& (sim_iter % 20 == 0)) {
			nrn_current[tid] = 5000;
		}
		// C4
		if (60 <= tid && tid <= 79
			&& (300 + shift_time_by_step) <= sim_iter
			&& sim_iter < (500 + shift_time_by_step)
			&& (sim_iter % 20 == 0)) {
			nrn_current[tid] = 5000;
		}
		// C5
		if (80 <= tid && tid <= 99
			&& (500 + shift_time_by_step) <= sim_iter
			&& sim_iter < (600 + shift_time_by_step)
			&& (sim_iter % 20 == 0)) {
			nrn_current[tid] = 5000;
		}

		if (1165 <= tid && tid <= 1184 && (sim_iter % 100 == 0)) {
			nrn_current[tid] = 5000;
		}

		// todo check with the real neurobiology mechanism
		// absolute refractory period : calculate V_m and U_m WITHOUT synaptic weight (nrn_current)
		// action potential : calculate V_m and U_m WITH synaptic weight (nrn_current)
		if (nrn_ref_time_timer[tid] > 0)
			nrn_current[tid] = 0;

		float V_old = old_v[tid];
		float U_old = old_u[tid];
		float I_current = nrn_current[tid];

		// ToDo check this with biological data (aprx)
		// the maximal value of input current (10 000 pA = 10 nA)
		if (I_current > 10000)
			I_current = 10000;
		if (I_current < -10000)
			I_current = -10000;

		// re-calculate V_m and U_m
		float V_m = V_old + SIM_STEP * (k * (V_old - V_rest) * (V_old - V_thld) - U_old + I_current) / C;
		float U_m = U_old + SIM_STEP * a * (b * (V_old - V_rest) - U_old);

		// set bottom border of the membrane potential
		if (V_m < c)
			V_m = c;
		// set top border of the membrane potential
		if (V_m >= V_thld)
			V_m = V_peak;

		// ToDo remove after debugging
		int index = sim_iter + tid * sim_time_in_step;
		voltage_recording[index] = V_m;

		// threshold crossing (spike)
		if (V_m >= V_thld) {
			// set spike status
			has_spike[tid] = true;
			// redefine V_old and U_old
			old_v[tid] = c;
			old_u[tid] += d;
			// set the refractory period
			nrn_ref_time_timer[tid] = nrn_ref_time[tid];
		} else {
			// redefine V_old and U_old
			old_v[tid] = V_m;
			old_u[tid] = U_m;
		}

		// wait all threads
		__syncthreads();

		// pointers to current neuronID synapses_delay_timer (decrease array calls)
		int *ptr_delay_timers = synapses_delay_timer[tid];
		// synapse updating loop
		for (int syn_id = 0; syn_id < synapses_number[tid]; syn_id++) {
			// add synaptic delay if neuron has spike
			if (has_spike[tid] && ptr_delay_timers[syn_id] == -1) {
				ptr_delay_timers[syn_id] = synapses_delay[tid][syn_id];
			}
			// if synaptic delay is zero it means the time when synapse increase I by synaptic weight
			if (ptr_delay_timers[syn_id] == 0) {
				// post neuron ID = synapses_post_nrn_id[tid][syn_id], thread-safe (!)
				atomicAdd(&nrn_current[ synapses_post_nrn_id[tid][syn_id] ], synapses_weight[tid][syn_id]);
				// make synapse timer a "free" for next spikes
				ptr_delay_timers[syn_id] = -1;
			}
			// update synapse delay timer
			if (ptr_delay_timers[syn_id] > 0) {
				ptr_delay_timers[syn_id]--;
			}
		} // end synapse updating loop

		// reset spike flag of the current neuron
		has_spike[tid] = false;

		// update currents of the neuron
		if (I_current != 0) {
			// decrease current potential for positive and negative current
			if (I_current > 0) nrn_current[tid] = I_current / 2;
			if (I_current < 0) nrn_current[tid] = I_current / 1.1f;
			// avoid the near value to 0
			if (I_current > 0 && I_current <= 1) nrn_current[tid] = 0;
			if (I_current <= 0 && I_current >= -1) nrn_current[tid] = 0;
		}

		// update the refractory period timer
		if (nrn_ref_time_timer[tid] > 0)
			nrn_ref_time_timer[tid]--;
	} // end of neuron stride loop
}

void connect_fixed_outdegree(Group pre_neurons, Group post_neurons, float syn_delay, float weight, int outdegree = syn_outdegree) {
	// connect neurons with uniform distribution and normal distributon for syn delay and weight
	weight *= (100 * 0.7);
	random_device rd;
	mt19937 gen(rd());	// Initialize pseudo-random number generator

	uniform_int_distribution<int> id_distr(post_neurons.id_start, post_neurons.id_end);
	normal_distribution<float> weight_distr(weight, 2);
	normal_distribution<float> delay_distr(syn_delay, 0.1);

	for (int pre_id = pre_neurons.id_start; pre_id <= pre_neurons.id_end; pre_id++) {
		for (int i = 0; i < outdegree; i++) {
			int rand_post_id = id_distr(gen);
			float syn_delay_dist = syn_delay;   // ToDo replace after tuning : delay_distr(gen);
			float syn_weight_dist = weight;     // ToDo replace after tuning : weight_distr(gen);
			metadatas.at(pre_id).push_back(SynapseMetadata(rand_post_id, syn_delay_dist, syn_weight_dist));
		}
	}

	printf("Connect %s with %s (1:%d). W=%.2f, D=%.1f\n",
		   pre_neurons.group_name.c_str(),
		   post_neurons.group_name.c_str(),
		   post_neurons.group_size,
		   weight,
		   syn_delay);
}

void init_extensor() {
	connect_fixed_outdegree(C3, inh_group3, 0.5, 15.0);
	connect_fixed_outdegree(C4, inh_group4, 0.5, 15.0);
	connect_fixed_outdegree(C5, inh_group5, 0.5, 15.0);

	connect_fixed_outdegree(inh_group3, G1_3, 2.8, 20.0);

	connect_fixed_outdegree(inh_group4, G1_3, 1.0, 20.0);
	connect_fixed_outdegree(inh_group4, G2_3, 1.0, 20.0);

	connect_fixed_outdegree(inh_group5, G1_3, 2.0, 20.0);
	connect_fixed_outdegree(inh_group5, G2_3, 1.0, 20.0);
	connect_fixed_outdegree(inh_group5, G3_3, 1.0, 20.0);
	connect_fixed_outdegree(inh_group5, G4_3, 1.0, 20.0);

	/// D1
	// input from sensory
	connect_fixed_outdegree(C1, D1_1, 1, 0.4);
	connect_fixed_outdegree(C1, D1_4, 1, 0.4);
	connect_fixed_outdegree(C2, D1_1, 1, 0.4);
	connect_fixed_outdegree(C2, D1_4, 1, 0.4);
	// input from EES
	connect_fixed_outdegree(EES, D1_1, 2, 10); // ST value (?)
	connect_fixed_outdegree(EES, D1_4, 2, 10); // ST value (?)
	// inner connectomes
	connect_fixed_outdegree(D1_1, D1_2, 1, 1.0);
	connect_fixed_outdegree(D1_1, D1_3, 1, 10.0);
	connect_fixed_outdegree(D1_2, D1_1, 1, 7.0);
	connect_fixed_outdegree(D1_2, D1_3, 1, 10.0);
	connect_fixed_outdegree(D1_3, D1_1, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D1_3, D1_2, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D1_4, D1_3, 3, -20 * INH_COEF);
	// output to
	connect_fixed_outdegree(D1_3, G1_1, 3, 8);
	connect_fixed_outdegree(D1_3, ees_group1, 1.0, 60);

	// EES group connectomes
	connect_fixed_outdegree(ees_group1, ees_group2, 1.0, 20.0);

	/// D2
	// input from Sensory
	connect_fixed_outdegree(C2, D2_1, 1, 0.8);
	connect_fixed_outdegree(C2, D2_4, 1, 0.8);
	connect_fixed_outdegree(C3, D2_1, 1, 0.8);
	connect_fixed_outdegree(C3, D2_4, 1, 0.8);
	// input from Group (1)
	connect_fixed_outdegree(ees_group1, D2_1, 1.7, 0.8);
	connect_fixed_outdegree(ees_group1, D2_4, 1.7, 1.0);
	// inner connectomes
	connect_fixed_outdegree(D2_1, D2_2, 1.0, 3.0);
	connect_fixed_outdegree(D2_1, D2_3, 1.0, 10.0);
	connect_fixed_outdegree(D2_2, D2_1, 1.0, 7.0);
	connect_fixed_outdegree(D2_2, D2_3, 1.0, 20.0);
	connect_fixed_outdegree(D2_3, D2_1, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D2_3, D2_2, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D2_4, D2_3, 2.0, -20 * INH_COEF);
	// output to generator
	connect_fixed_outdegree(D2_3, G2_1, 1.0, 8);

	// EES group connectomes
	connect_fixed_outdegree(ees_group2, ees_group3, 1.0, 20.0);

	/// D3
	// input from Sensory
	connect_fixed_outdegree(C3, D3_1, 1, 0.5);
	connect_fixed_outdegree(C3, D3_4, 1, 0.5);
	connect_fixed_outdegree(C4, D3_1, 1, 0.5);
	connect_fixed_outdegree(C4, D3_4, 1, 0.5);
	// input from Group (2)
	connect_fixed_outdegree(ees_group2, D3_1, 1, 1.2);
	connect_fixed_outdegree(ees_group2, D3_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D3_1, D3_2, 1.0, 3.0);
	connect_fixed_outdegree(D3_1, D3_3, 1.0, 10.0);
	connect_fixed_outdegree(D3_2, D3_1, 1.0, 7.0);
	connect_fixed_outdegree(D3_2, D3_3, 1.0, 20.0);
	connect_fixed_outdegree(D3_3, D3_1, 1.0, -10 * INH_COEF);
	connect_fixed_outdegree(D3_3, D3_2, 1.0, -10 * INH_COEF);
	connect_fixed_outdegree(D3_4, D3_3, 2.0, -10 * INH_COEF);
	// output to generator
	connect_fixed_outdegree(D3_3, G3_1, 1, 25.0);
	// suppression of the generator
	connect_fixed_outdegree(D3_3, G1_3, 1.5, 30.0);

	// EES group connectomes
	connect_fixed_outdegree(ees_group3, ees_group4, 2.0, 20.0);

	/// D4
	// input from Sensory
	connect_fixed_outdegree(C4, D4_1, 1, 0.5);
	connect_fixed_outdegree(C4, D4_4, 1, 0.5);
	connect_fixed_outdegree(C5, D4_1, 1, 0.5);
	connect_fixed_outdegree(C5, D4_4, 1, 0.5);
	// input from Group (3)
	connect_fixed_outdegree(ees_group3, D4_1, 1, 1.2);
	connect_fixed_outdegree(ees_group3, D4_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D4_1, D4_2, 1.0, 3.0);
	connect_fixed_outdegree(D4_1, D4_3, 1.0, 10.0);
	connect_fixed_outdegree(D4_2, D4_1, 1.0, 7.0);
	connect_fixed_outdegree(D4_2, D4_3, 1.0, 20.0);
	connect_fixed_outdegree(D4_3, D4_1, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_3, D4_2, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_4, D4_3, 2.0, -20 * INH_COEF);
	// output to the generator
	connect_fixed_outdegree(D4_3, G4_1, 3.0, 20.0);
	// suppression of the generator
	connect_fixed_outdegree(D4_3, G2_3, 1.0, 30.0);

	/// D5
	// input from Sensory
	connect_fixed_outdegree(C5, D5_1, 1, 0.5);
	connect_fixed_outdegree(C5, D5_4, 1, 0.5);
	// input from Group (4)
	connect_fixed_outdegree(ees_group4, D5_1, 1.0, 1.1);
	connect_fixed_outdegree(ees_group4, D5_4, 1.0, 1.0);
	// inner connectomes
	connect_fixed_outdegree(D5_1, D5_2, 1.0, 3.0);
	connect_fixed_outdegree(D5_1, D5_3, 1.0, 15.0);
	connect_fixed_outdegree(D5_2, D5_1, 1.0, 7.0);
	connect_fixed_outdegree(D5_2, D5_3, 1.0, 20.0);
	connect_fixed_outdegree(D5_3, D5_1, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D5_3, D5_2, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D5_4, D5_3, 2.5, -20 * INH_COEF);
	// output to the generator
	connect_fixed_outdegree(D5_3, G5_1, 3, 8.0);
	// suppression of the genearator
	connect_fixed_outdegree(D5_3, G1_3, 1.0, 30.0);
	connect_fixed_outdegree(D5_3, G2_3, 1.0, 30.0);
	connect_fixed_outdegree(D5_3, G3_3, 1.0, 30.0);
	connect_fixed_outdegree(D5_3, G4_3, 1.0, 30.0);

	/// G1
	// inner connectomes
	connect_fixed_outdegree(G1_1, G1_2, 1.0, 10.0);
	connect_fixed_outdegree(G1_1, G1_3, 1.0, 15.0);
	connect_fixed_outdegree(G1_2, G1_1, 1.0, 10.0);
	connect_fixed_outdegree(G1_2, G1_3, 1.0, 15.0);
	connect_fixed_outdegree(G1_3, G1_1, 0.7, -70 * INH_COEF);
	connect_fixed_outdegree(G1_3, G1_2, 0.7, -70 * INH_COEF);
	// output to IP_E
	connect_fixed_outdegree(G1_1, IP_E, 3, 25.0);
	connect_fixed_outdegree(G1_1, IP_E, 3, 25.0);

	/// G2
	// inner connectomes
	connect_fixed_outdegree(G2_1, G2_2, 1.0, 10.0);
	connect_fixed_outdegree(G2_1, G2_3, 1.0, 20.0);
	connect_fixed_outdegree(G2_2, G2_1, 1.0, 10.0);
	connect_fixed_outdegree(G2_2, G2_3, 1.0, 20.0);
	connect_fixed_outdegree(G2_3, G2_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G2_3, G2_2, 0.5, -30 * INH_COEF);
	// output to IP_E
	connect_fixed_outdegree(G2_1, IP_E, 1.0, 65.0);
	connect_fixed_outdegree(G2_2, IP_E, 1.0, 65.0);

	/// G3
	// inner connectomes
	connect_fixed_outdegree(G3_1, G3_2, 1.0, 14.0);
	connect_fixed_outdegree(G3_1, G3_3, 1.0, 20.0);
	connect_fixed_outdegree(G3_2, G3_1, 1.0, 12.0);
	connect_fixed_outdegree(G3_2, G3_3, 1.0, 20.0);
	connect_fixed_outdegree(G3_3, G3_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G3_3, G3_2, 0.5, -30 * INH_COEF);
	// output to IP_E
	connect_fixed_outdegree(G3_1, IP_E, 2, 25.0);
	connect_fixed_outdegree(G3_1, IP_E, 2, 25.0);

	/// G4
	// inner connectomes
	connect_fixed_outdegree(G4_1, G4_2, 1.0, 10.0);
	connect_fixed_outdegree(G4_1, G4_3, 1.0, 10.0);
	connect_fixed_outdegree(G4_2, G4_1, 1.0, 5.0);
	connect_fixed_outdegree(G4_2, G4_3, 1.0, 10.0);
	connect_fixed_outdegree(G4_3, G4_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G4_3, G4_2, 0.5, -30 * INH_COEF);
	// output to IP_E
	connect_fixed_outdegree(G4_1, IP_E, 1.0, 17.0);
	connect_fixed_outdegree(G4_1, IP_E, 1.0, 17.0);

	/// G5
	// inner connectomes
	connect_fixed_outdegree(G5_1, G5_2, 1.0, 7.0);
	connect_fixed_outdegree(G5_1, G5_3, 1.0, 10.0);
	connect_fixed_outdegree(G5_2, G5_1, 1.0, 7.0);
	connect_fixed_outdegree(G5_2, G5_3, 1.0, 10.0);
	connect_fixed_outdegree(G5_3, G5_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G5_3, G5_2, 0.5, -30 * INH_COEF);
	// output to IP_E
	connect_fixed_outdegree(G5_1, IP_E, 2, 20.0);
	connect_fixed_outdegree(G5_1, IP_E, 2, 20.0);

	connect_fixed_outdegree(IP_E, MP_E, 1, 11);
	connect_fixed_outdegree(EES, MP_E, 2, 50);
	connect_fixed_outdegree(Ia, MP_E, 1, 1);
}

void save_result(int test_index,
				 float* voltage_recording,
				 int neurons_number) {
	// save results for each neuron (voltage/current/spikes)
	char cwd[256];
	ofstream myfile;

	getcwd(cwd, sizeof(cwd));
	printf("[Test #%d] Save results to: %s \n", test_index, cwd);

	string new_name = "/volt.dat";
	myfile.open(cwd + new_name);

	for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++){
		myfile << nrn_id << " ";
		for(int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++)
			myfile << voltage_recording[sim_iter + nrn_id * sim_time_in_step] << " ";
		myfile << "\n";
	}

	myfile.close();
}

template <typename type>
void memcpyHtD(type* gpu, type* host, int size) {
	hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice);
}

template <typename type>
void memcpyDtH(type* host, type* gpu, int size) {
	hipMemcpy(host, gpu, sizeof(type) * size, hipMemcpyDeviceToHost);
}

template <typename type>
unsigned int datasize(int size) {
	return sizeof(type) * size;
}

template <typename type>
void init_array(type *array, int size, type value){
	for(int i = 0; i < size; i++)
		array[i] = value;
}

__host__
void simulate(int test_index) {
	int neurons_number = static_cast<int>(metadatas.size());

	float* gpu_old_v;
	float* gpu_old_u;
	int* gpu_nrn_ref_time;
	int* gpu_nrn_ref_timer;
	bool* gpu_has_spike;
	float* gpu_nrn_current;
	int* gpu_synapses_number;

	float* gpu_voltage_recording;

	int synapses_number[neurons_number];

	float old_v[neurons_number];
	init_array<float>(old_v, neurons_number, V_rest);

	float old_u[neurons_number];
	init_array<float>(old_u, neurons_number, 0);

	int nrn_ref_time[neurons_number];
	init_array<int>(nrn_ref_time, neurons_number, ms_to_step(3.0));

	int nrn_ref_timer[neurons_number];
	init_array<int>(nrn_ref_timer, neurons_number, -1);

	bool has_spike[neurons_number];
	init_array<bool>(has_spike, neurons_number, false);

	float nrn_current[neurons_number];
	init_array<float>(nrn_current, neurons_number, 0);

	float* voltage_recording = (float *)malloc(datasize<float *>(neurons_number * sim_time_in_step));
	init_array<float>(voltage_recording, neurons_number * sim_time_in_step, -72);

	// init connectomes
	init_extensor();

	int **gpu_synapses_post_nrn_id, **synapses_post_nrn_id = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay, **synapses_delay = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay_timer, **synapses_delay_timer = (int **)malloc(datasize<int* >(neurons_number));
	float **gpu_synapses_weight, **synapses_weight = (float **)malloc(datasize<float* >(neurons_number));

	// fill arrays of synapses
	for(int neuron_id = 0; neuron_id < neurons_number; neuron_id++) {
		int syn_count = static_cast<int>(metadatas.at(neuron_id).size());

		int tmp_synapses_post_nrn_id[syn_count];
		int tmp_synapses_delay[syn_count];
		int tmp_synapses_delay_timer[syn_count];
		float tmp_synapses_weight[syn_count];

		int syn_id = 0;
		for(SynapseMetadata metadata : metadatas.at(neuron_id)) {
			tmp_synapses_post_nrn_id[syn_id] = metadata.post_id;
			tmp_synapses_delay[syn_id] = metadata.synapse_delay;
			tmp_synapses_delay_timer[syn_id] = -1;
			tmp_synapses_weight[syn_id] = metadata.synapse_weight;
			syn_id++;
		}

		synapses_number[neuron_id] = syn_count;

		hipMalloc((void**)&synapses_post_nrn_id[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay_timer[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_weight[neuron_id], datasize<float>(syn_count));

		hipMemcpy(synapses_post_nrn_id[neuron_id], &tmp_synapses_post_nrn_id, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay[neuron_id], &tmp_synapses_delay, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay_timer[neuron_id], &tmp_synapses_delay_timer, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_weight[neuron_id], &tmp_synapses_weight, datasize<float>(syn_count), hipMemcpyHostToDevice);
	}

	hipMalloc((void ***)&gpu_synapses_post_nrn_id, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_post_nrn_id, synapses_post_nrn_id, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay, synapses_delay, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay_timer, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay_timer, synapses_delay_timer, neurons_number);

	hipMalloc((void ***)&gpu_synapses_weight, datasize<float *>(neurons_number));
	memcpyHtD<float *>(gpu_synapses_weight, synapses_weight, neurons_number);

	hipMalloc(&gpu_old_v, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_old_v, old_v, neurons_number);

	hipMalloc(&gpu_old_u, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_old_u, old_u, neurons_number);

	hipMalloc(&gpu_has_spike, datasize<bool>(neurons_number));
	memcpyHtD<bool>(gpu_has_spike, has_spike, neurons_number);

	hipMalloc(&gpu_nrn_ref_time, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_time, nrn_ref_time, neurons_number);

	hipMalloc(&gpu_nrn_ref_timer, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_timer, nrn_ref_timer, neurons_number);

	hipMalloc(&gpu_nrn_current, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_nrn_current, nrn_current, neurons_number);

	hipMalloc(&gpu_synapses_number, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_synapses_number, synapses_number, neurons_number);

	hipMalloc(&gpu_voltage_recording, datasize<float>(neurons_number * sim_time_in_step));
	memcpyHtD<float>(gpu_voltage_recording, voltage_recording, neurons_number * sim_time_in_step);

	int threads_per_block = 1024;
	int num_blocks = 1; //neurons_number / threads_per_block + 1;

	printf("Size of network: %i \n", neurons_number);
	printf("Start GPU with %d threads x %d blocks (Total: %d threads) \n",
		   threads_per_block, num_blocks, threads_per_block * num_blocks);

	int shift_time_by_step = 0;
	int activated_C_ = 0;
	int local_iter = 0;

	// measure GPU ellapsed time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	chrono::time_point<chrono::system_clock> iter_t_start, iter_t_end, simulation_t_start, simulation_t_end;
	chrono::duration<double> elapsed_time_per_iter[sim_time_in_step];
	chrono::duration<double> waited_time_per_iter[sim_time_in_step];

	std::thread timer_thread;

	simulation_t_start = chrono::system_clock::now();

	// GPU max T per step (4000 steps) <= 250 µm (0.25 ms)
	for (int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++) {
		// start measure time
		iter_t_start = chrono::system_clock::now();

		// if flexor C0 activated, find the end of it and change to C1
		if (activated_C_ == 0) {
			if (local_iter != 0 && local_iter % steps_activation_C0 == 0) {
				activated_C_ = 1; // change to C1
				local_iter = 0;   // reset local time iterator
				shift_time_by_step += steps_activation_C0;  // add constant 125 ms
			}
		// if extensor C1 activated, find the end of it and change to C0
		} else {
			if (local_iter != 0 && local_iter % steps_activation_C1 == 0) {
				activated_C_ = 0; // change to C0
				local_iter = 0;   // reset local time iterator
				shift_time_by_step += steps_activation_C1;  // add time equal to n_layers * 25 ms
			}
		}

		sim_kernel<<<num_blocks, threads_per_block>>>(gpu_old_v,
				gpu_old_u,
				gpu_nrn_current,
				gpu_nrn_ref_time,
				gpu_nrn_ref_timer,
				gpu_synapses_number,
				gpu_has_spike,
				gpu_synapses_post_nrn_id,
				gpu_synapses_delay,
				gpu_synapses_delay_timer,
				gpu_synapses_weight,
				neurons_number,
				gpu_voltage_recording,
				activated_C_,
				shift_time_by_step,
				sim_iter);
		local_iter++;

		// ToDo here will be the DAC functionality
		std::this_thread::sleep_for( std::chrono::microseconds(100) );

		// stop measure time
		iter_t_end = std::chrono::system_clock::now();
		// save time difference
		elapsed_time_per_iter[sim_iter] = chrono::duration_cast<chrono::microseconds>(iter_t_end - iter_t_start);

		if (elapsed_time_per_iter[sim_iter].count() * 1000000 < 250) {
			auto waited = std::chrono::microseconds( (int)(250 - elapsed_time_per_iter[sim_iter].count() * 1000000) );
			waited_time_per_iter[sim_iter] = std::chrono::microseconds(waited);
			std::this_thread::sleep_for(waited);
		}

	} // end of simulation iteration loop

	simulation_t_end = chrono::system_clock::now();

	double sum = 0;
	double wai = 0;

	for (int i = 0; i < sim_time_in_step; i++) {
		sum += elapsed_time_per_iter[i].count();
		wai += waited_time_per_iter[i].count();
		cout << i << " " <<  elapsed_time_per_iter[i].count() << " µs, w=" << waited_time_per_iter[i].count() << endl;
	}

	auto sim_time_diff = chrono::duration_cast<chrono::milliseconds>(simulation_t_end - simulation_t_start).count();
	printf("Elapsed %li ms (measured), used %.2f ms, waited %.2f, T_sim = %.2f ms\n", sim_time_diff, sum * 1000, wai * 1000, T_sim);
	printf("%s x%f\n", (double)(T_sim / sim_time_diff) > 1? "faster" : "slower", T_sim / sim_time_diff);
//	printf("Ellapsed time: %fs. Realtime factor: x%f (%s than realtime)\n",
//		   t, realtime_factor, realtime_factor > 1? "faster":"slower");

	// copy neurons/synapses array to the HOST
	memcpyDtH<float>(voltage_recording, gpu_voltage_recording, neurons_number * sim_time_in_step);

	// tell the CPU to halt further processing until the CUDA kernel has finished doing its business
	hipDeviceSynchronize();

//	save_result(test_index, voltage_recording, neurons_number);

	hipDeviceReset();
}

int main() {
	simulate(0);

	return 0;
}