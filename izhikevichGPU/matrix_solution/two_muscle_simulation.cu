#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <vector>
#include <ctime>
#include <stdexcept>
#include <random>
// for file writing
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
// my classes
#include "Group.cpp"
// jetbrains cuda
#ifdef __JETBRAINS_IDE__
	#define __host__
	#define __device__
	#define __global__
	#define __shared__
#endif

using namespace std;

unsigned int global_id = 0;

const int EES_FREQ = 40;
const int SEN_FREQ = 200;
const float INH_COEF = 0.0f;

// stuff variable
const unsigned int syn_outdegree = 27;
const unsigned int neurons_in_ip = 196;
const unsigned int neurons_in_moto = 169;
const unsigned int neurons_in_group = 20;
const unsigned int neurons_in_afferent = 60;
// 6 CMS = 125	[ms] 30
// 15 CMS = 50	[ms] 15
// 21 CMS = 25	[ms] 6
const int skin_stim_time = 25;
const int slices_number = 6;

const float T_sim = 275;
const float sim_step = 0.25;

__host__
int ms_to_step(float ms) { return (int)(ms / sim_step); }

const int sensory_spike_each_step = (int)(1000 / SEN_FREQ / sim_step);
const int ees_spike_each_step = (int)(1000 / EES_FREQ / sim_step);

const int C1_begin_spiking_time = (int)(0.1 / sim_step);
const int C2_begin_spiking_time = (int)(skin_stim_time / sim_step);
const int C3_begin_spiking_time = (int)(2 * skin_stim_time / sim_step);
const int C4_begin_spiking_time = (int)(3 * skin_stim_time / sim_step);
const int C5_begin_spiking_time = (int)(5 * skin_stim_time / sim_step);

const int C1_end_spiking_time = (int)(skin_stim_time / sim_step);
const int C2_end_spiking_time = (int)(2 * skin_stim_time / sim_step);
const int C3_end_spiking_time = (int)(3 * skin_stim_time / sim_step);
const int C4_end_spiking_time = (int)(5 * skin_stim_time / sim_step);
const int C5_end_spiking_time = (int)(6 * skin_stim_time / sim_step);

const unsigned int steps_activation_C0 = (unsigned int)(skin_stim_time * 5 / sim_step);
const unsigned int steps_activation_C1 = (unsigned int)(skin_stim_time * slices_number / sim_step);

const unsigned int sim_time_in_step = (unsigned int)(T_sim / sim_step);

struct SynapseMetadata{
	// struct for human-readable initialization of connectomes
	int post_id;
	int synapse_delay;
	float synapse_weight;

	SynapseMetadata() = default;
	SynapseMetadata(int post_id, float synapse_delay, float synapse_weight){
		this->post_id = post_id;
		this->synapse_delay = static_cast<int>(synapse_delay * (1 / sim_step) + 0.5); // round
		this->synapse_weight = synapse_weight;
	}
};

Group form_group(string group_name, int nrns_in_group = neurons_in_group) {
	// form structs of neurons global ID and groups name
	Group group = Group();

	group.group_name = group_name;
	group.id_start = global_id;
	group.id_end = global_id + nrns_in_group - 1;
	group.group_size = nrns_in_group;

	global_id += nrns_in_group;

	printf("Formed %s IDs [%d ... %d] = %d\n", group_name.c_str(), global_id - nrns_in_group, global_id - 1, nrns_in_group);

	return group;
}

/* Nodes with changable connectomes
C=0 :: disable for neurons 0 <= tid <= 39 and their last 27 synapses. Slice as [:-27]
[D2_3, D4_3]

C=1 :: disable for neurons 0 <= tid <= 219 and their first 27 synapses. Slice as [27:]
[D2_3, D4_3, D1_3, G2_1, G2_2, G3_1, G3_2, G4_1, G4_2, G5_1, G5_2] */

// Form neuron groups
// At first init nodes with changable connectomes to reduce "and" operators at synapse checking (by tid)
// C=0 group
Group D2_3 = form_group("D2_3");	//  D2_3 IDs [0 ... 19]
Group D4_3 = form_group("D4_3");	//  D4_3 IDs [20 ... 39]

// C=1 group
Group D1_3 = form_group("D1_3");	// D1_3 IDs [40 ... 59]
Group G2_1 = form_group("G2_1");
Group G2_2 = form_group("G2_2");
Group G3_1 = form_group("G3_1");
Group G3_2 = form_group("G3_2");
Group G4_1 = form_group("G4_1");
Group G4_2 = form_group("G4_2");
Group G5_1 = form_group("G5_1");
Group G5_2 = form_group("G5_2");	// G5_2 IDs [200 ... 219]

// groups of neurons without changable synapses

// groups of neurons with generators
Group C1 = form_group("C1");
Group C2 = form_group("C2");
Group C3 = form_group("C3");
Group C4 = form_group("C4");
Group C5 = form_group("C5");
Group EES = form_group("EES");

// just groups of neurons
Group D1_1 = form_group("D1_1");
Group D1_2 = form_group("D1_2");
Group D1_4 = form_group("D1_4");

Group D2_1 = form_group("D2_1");
Group D2_2 = form_group("D2_2");
Group D2_4 = form_group("D2_4");

Group D3_1 = form_group("D3_1");
Group D3_2 = form_group("D3_2");
Group D3_3 = form_group("D3_3");
Group D3_4 = form_group("D3_4");

Group D4_1 = form_group("D4_1");
Group D4_2 = form_group("D4_2");
Group D4_4 = form_group("D4_4");

Group D5_1 = form_group("D5_1");
Group D5_2 = form_group("D5_2");
Group D5_3 = form_group("D5_3");
Group D5_4 = form_group("D5_4");

Group G1_1 = form_group("G1_1");
Group G1_2 = form_group("G1_2");
Group G1_3 = form_group("G1_3");

Group G2_3 = form_group("G2_3");

Group G3_3 = form_group("G3_3");

Group G4_3 = form_group("G4_3");

Group G5_3 = form_group("G5_3");

Group IP_E = form_group("IP_E", neurons_in_ip);
Group IP_F = form_group("IP_F", neurons_in_ip);

Group MP_E = form_group("MP_E", neurons_in_moto);
Group MP_F = form_group("MP_F", neurons_in_moto);

Group Ia_Extensor = form_group("Ia_Extensor", neurons_in_afferent);
Group Ia_Flexor = form_group("Ia_Flexor", neurons_in_afferent);

Group inh_group3 = form_group("inh_group3");
Group inh_group4 = form_group("inh_group4");
Group inh_group5 = form_group("inh_group5");

Group ees_group1 = form_group("ees_group1");
Group ees_group2 = form_group("ees_group2");
Group ees_group3 = form_group("ees_group3");
Group ees_group4 = form_group("ees_group4");

Group R_E = form_group("R_E");
Group R_F = form_group("R_F");

Group Ia_E = form_group("Ia_E");
Group Ia_F = form_group("Ia_F");
Group Ib_E = form_group("Ib_E");
Group Ib_F = form_group("Ib_F");

//Group С_0 = form_group("С_0");
//Group С_1 = form_group("С_1");

// Global vectors of SynapseMetadata of synapses for each neuron
vector<vector<SynapseMetadata>> metadatas(global_id, vector<SynapseMetadata>());

// Parameters (const)
const float C = 100;        // [pF] membrane capacitance
const float V_rest = -72;   // [mV] resting membrane potential
const float V_thld = -55;   // [mV] spike threshold
const float k = 0.7;        // [pA * mV-1] constant ("1/R")
const float a = 0.02;       // [ms-1] time scale of the recovery variable U_m. Higher a, the quicker recovery
const float b = 0.2;        // [pA * mV-1] sensitivity of U_m to the sub-threshold fluctuations of the V_m
const float c = -80;        // [mV] after-spike reset value of V_m
const float d = 6;          // [pA] after-spike reset value of U_m
const float V_peak = 35;    // [mV] spike cutoff value

const float sim_step_C_const = sim_step / C;
const float sim_step_a_const = sim_step * a;

__global__
void sim_kernel(float* old_v,
                float* old_u,
                float* nrn_current,
                int* nrn_ref_time,
                int* nrn_ref_time_timer,
                int* synapses_number,
                bool* has_spike,
                int** synapses_post_nrn_id,
                int** synapses_delay,
                int** synapses_delay_timer,
                float** synapses_weight,
                unsigned int nrn_size,
                // ToDo remove after debugging
                float* voltage_recording,
                float* current_recording,
                int* spike_recording) {
	// FixMe: hidden bug, but will work perfect if number of spikes will be lower than sim_step_time / 2 (usually)
	// FixMe: explanation -- each thread has local variable, but here is stride loop. So one thread do at least 2 job
	int local_spike_array_iter = 0;

	// get id of the thread
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

	// activated_C_ 0 - at flexor (TA)
	// activated_C_ 1 - at extensor (MG)
	__shared__ short activated_C_;
	__shared__ short shift_time_by_step ;
	__shared__ short decrease_lvl_Ia_spikes;
	__shared__ bool sensory_spike_flag;

	if(thread_id == 0) {
		activated_C_ = 0;
		shift_time_by_step = 0;
		decrease_lvl_Ia_spikes = 1;
		sensory_spike_flag = false;
	}

	int master_local_iter = 0;

	// the main simulation loop
	for (int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++) {
		// wait all threads
		__syncthreads();

		// mechanism of changing C0 and C1
		if(thread_id == 0) {
			// if flexor C0 activated, find the end of it and change to C1
			if (activated_C_ == 0) {
				if (master_local_iter != 0 && master_local_iter % steps_activation_C0 == 0) {
					activated_C_ = 1;
					master_local_iter = 0;

					shift_time_by_step += steps_activation_C0; // add const 125 ms
				}
			// if extensor C1 activated, find the end of it and change to C0
			} else {
				if (master_local_iter != 0 && master_local_iter % steps_activation_C1 == 0) {
					activated_C_ = 0;
					master_local_iter = 0;

					shift_time_by_step += steps_activation_C1; // add layers * 25 to the shift
				}
			}

			// printf("step %d [local %d] (%.2f ms) with C%d \n", sim_iter, master_local_iter, sim_iter * sim_step, activated_C_);
			master_local_iter++;

			// reset variable
			decrease_lvl_Ia_spikes = 1;
			sensory_spike_flag = sim_iter % sensory_spike_each_step == 0;
		}

		// wait all threads
		__syncthreads();

		// neuron (tid = neuron id) stride loop (0, 1024, 1, 1025 ...)
		for (int tid = thread_id; tid < nrn_size; tid += blockDim.x * gridDim.x) {
			// activate only sensory C1-C5 and control Ia afferent spikes
			if (220 <= tid && tid <= 319) {
				int shifted_sim_iter = sim_iter - shift_time_by_step;
				// C1
				if (220 <= tid && tid <= 239 && (C1_begin_spiking_time <= shifted_sim_iter) &&
					(shifted_sim_iter < C1_end_spiking_time)) {
					if (tid == 220) {
						decrease_lvl_Ia_spikes = 3;
					}
					if (activated_C_ == 1 && sensory_spike_flag) {
						nrn_current[tid] = 5000; // set spike state
					}
				} else {
					// C2
					if (240 <= tid && tid <= 259 && (C2_begin_spiking_time <= shifted_sim_iter) &&
						(shifted_sim_iter < C2_end_spiking_time)) {
						if (tid == 240) {
							decrease_lvl_Ia_spikes = 2;
						}
						if (activated_C_ == 1 && sensory_spike_flag) {
							nrn_current[tid] = 5000; // set spike state
						}
					} else {
						// C3
						if (260 <= tid && tid <= 279 && (C3_begin_spiking_time <= shifted_sim_iter) &&
							(shifted_sim_iter < C3_end_spiking_time)) {
							if (tid == 260) {
								decrease_lvl_Ia_spikes = 1;
							}
							if (activated_C_ == 1 && sensory_spike_flag) {
								nrn_current[tid] = 5000; // set spike state
							}
						} else {
							// C4
							if (280 <= tid && tid <= 299 && (C4_begin_spiking_time <= shifted_sim_iter) &&
								(shifted_sim_iter < C4_end_spiking_time)) {
								if (tid == 280) {
									decrease_lvl_Ia_spikes = 2;
								}
								if (activated_C_ == 1 && sensory_spike_flag) {
									nrn_current[tid] = 5000; // set spike state
								}
							} else {
								// C5
								if (300 <= tid && tid <= 319 && (C5_begin_spiking_time <= shifted_sim_iter) &&
									(shifted_sim_iter < C5_end_spiking_time)) {
									if (tid == 300) {
										decrease_lvl_Ia_spikes = 3;
									}
									if (activated_C_ == 1 && sensory_spike_flag) {
										nrn_current[tid] = 5000; // set spike state
									}
								}
							}
						}
					}
				}
			}

			__syncthreads();

			// generating spikes for EES
			if (320 <= tid && tid <= 339 && (sim_iter % ees_spike_each_step == 0)) {
				nrn_current[tid] = 5000;
			}

			// Ia IDs [1550 ... 1669], control spike number of Ia afferent by resetting neuron current
			if (1550 <= tid && tid <= 1669) {
				// rule for the 2nd level
				if (decrease_lvl_Ia_spikes == 2 && tid % 3 == 0) {
					// reset current of 1/3 of neurons
					nrn_current[tid] = 0;
				} else {
					// rule for the 3rd level
					if (decrease_lvl_Ia_spikes == 3 && tid % 2 == 0) {
						// reset current of 1/2 of neurons
						nrn_current[tid] = 0;
					}
				}
			}

			// inhibit IP_E IDs [820 ... 1015] by C0
			if (activated_C_ == 0 && 820 <= tid && tid <= 1015) {
				nrn_current[tid] *= (1 - INH_COEF);
			}
			// inhibit IP_F IDs [1016 ... 1211] by C1
			if (activated_C_ == 1 && 1016 <= tid && tid <= 1211){
				nrn_current[tid] *= (1 - INH_COEF);
			}
			// inhibit Ia_Extensor IDs [1550 ... 1609] by C0
			if (activated_C_ == 0 && 1550 <= tid && tid <= 1609) {
				nrn_current[tid] *= (1 - INH_COEF);
			}
			// inhibit Ia_Flexor IDs [1610 ... 1669] by C1
			if (activated_C_ == 1 && 1610 <= tid && tid <= 1669) {
				nrn_current[tid] *= (1 - INH_COEF);
			}

			// todo check with the real neurobiology mechanism
			// absolute refractory period : calculate V_m and U_m WITHOUT synaptic weight (nrn_current)
			// action potential : calculate V_m and U_m WITH synaptic weight (nrn_current)
			if (nrn_ref_time_timer[tid] > 0)
				nrn_current[tid] = 0;

			float V_old = old_v[tid];
			float U_old = old_u[tid];
			float I_current = nrn_current[tid];

			// ToDo check this with biological data (aprx)
			// the maximal value of input current (10 000 pA = 10 nA)
			if (I_current > 10000)
				I_current = 10000;
			if (I_current < -10000)
				I_current = -10000;

			// re-calculate V_m and U_m
			float V_m = V_old + sim_step_C_const * (k * (V_old - V_rest) * (V_old - V_thld) - U_old + I_current);
			float U_m = U_old + sim_step_a_const * (b * (V_old - V_rest) - U_old);

			// set bottom border of the membrane potential
			if (V_m < c)
				V_m = c;
			// set top border of the membrane potential
			if (V_m >= V_thld)
				V_m = V_peak;

			// record the membrane potential value every iter step if neuron has multimeter
//			if (1381 <= tid && tid <= 1549) {
//				atomicAdd(&multimeter_result[sim_iter], V_m);
//			}

			// ToDo remove after debugging
			int index = sim_iter + tid * sim_time_in_step;
			voltage_recording[index] = V_m;
			current_recording[index] = I_current;

			// threshold crossing (spike)
			if (V_m >= V_thld) {
				// set spike status
				has_spike[tid] = true;
				// redefine V_old and U_old
				old_v[tid] = c;
				old_u[tid] += d;
				// set the refractory period
				nrn_ref_time_timer[tid] = nrn_ref_time[tid];
				// ToDo remove after debugging
				spike_recording[local_spike_array_iter + tid * sim_time_in_step] = sim_iter;
				local_spike_array_iter++;
			} else {
				// redefine V_old and U_old
				old_v[tid] = V_m;
				old_u[tid] = U_m;
			}

			// pointers to current neuronID synapses_delay_timer (decrease array calls)
			int *ptr_delay_timers = synapses_delay_timer[tid];

			// init basic synapse ids
			int syn_id_begin = 0;
			int syn_id_end = synapses_number[tid];

			// C=0 -- "slice" as [:-27] -- skip the last 27 synapses because they must be inhibited
			if(activated_C_ == 0 && 0 <= tid && tid <= 39){
				syn_id_end -= 27;
			}
			// C=1 -- "slice" as [54:] -- skip the first 54 synapses because they must be inhibited
			if(activated_C_ == 1 && 0 <= tid && tid <= 219){
				syn_id_begin = 54; // 27 * 2
			}

			// synapse updating loop (with formed begin/end borders)
			for (int syn_id = syn_id_begin; syn_id < syn_id_end; syn_id++) {
				// add synaptic delay if neuron has spike
				if (has_spike[tid] && ptr_delay_timers[syn_id] == -1) {
					ptr_delay_timers[syn_id] = synapses_delay[tid][syn_id];
				}
				// if synaptic delay is zero it means the time when synapse increase I by synaptic weight
				if (ptr_delay_timers[syn_id] == 0) {
					// post neuron ID = synapses_post_nrn_id[tid][syn_id], thread-safe (!)
					atomicAdd(&nrn_current[ synapses_post_nrn_id[tid][syn_id] ], synapses_weight[tid][syn_id]);
					// make synapse timer a "free" for next spikes
					ptr_delay_timers[syn_id] = -1;
				}
				// update synapse delay timer
				if (ptr_delay_timers[syn_id] > 0) {
					ptr_delay_timers[syn_id]--;
				}
			} // end synapse updating loop

			// reset spike flag of the current neuron
			has_spike[tid] = false;

			// update currents of the neuron
			if (I_current != 0) {
				// decrease current potential for positive and negative current
				if (I_current > 0) nrn_current[tid] = I_current / 2;
				if (I_current < 0) nrn_current[tid] = I_current / 1.1f;
				// avoid the near value to 0
				if (0 < I_current && I_current <= 1) nrn_current[tid] = 0;
				if (-1 <= I_current && I_current <= 0) nrn_current[tid] = 0;
			}

			// update the refractory period timer
			if (nrn_ref_time_timer[tid] > 0)
				nrn_ref_time_timer[tid]--;
		} // end of neuron stride loop

		// wait all threads
		__syncthreads();

	} // end of sim iteration
}

void connect_fixed_outdegree(Group pre_neurons, Group post_neurons,
                             float syn_delay, float weight, int outdegree = syn_outdegree) {
	// connect neurons with uniform distribution and normal distributon for syn delay and weight
	weight *= (100 * 0.7);

	random_device rd;
	mt19937 gen(rd());	// Initialize pseudo-random number generator

	uniform_int_distribution<int> id_distr(post_neurons.id_start, post_neurons.id_end);
	normal_distribution<float> delay_distr(syn_delay, syn_delay / 10);
	normal_distribution<float> weight_distr(weight, weight / 10);

	#ifdef DEBUG
	printf("pre group %s (%d, %d) to post %s (%d, %d)\n",
	       pre_neurons.group_name.c_str(),
	       pre_neurons.id_start,
	       pre_neurons.id_end,
	       post_neurons.group_name.c_str(),
	       post_neurons.id_start,
	       post_neurons.id_end);
	#endif

	for (int pre_id = pre_neurons.id_start; pre_id <= pre_neurons.id_end; pre_id++) {
		for (int i = 0; i < outdegree; i++) {
			int rand_post_id = id_distr(gen);
			float syn_delay_dist = syn_delay; //delay_distr(gen);
			float syn_weight_dist = weight_distr(gen);
			#ifdef DEBUG
			printf("weight %f (%f), delay %f (%f) \n",
					syn_weight_dist, weight, syn_delay_dist, syn_delay);
			#endif
			metadatas.at(pre_id).push_back(SynapseMetadata(rand_post_id, syn_delay_dist, syn_weight_dist));
		}
	}

	printf("Connect %s with %s (1:%d). W=%.2f, D=%.1f\n", pre_neurons.group_name.c_str(),
	                                                      post_neurons.group_name.c_str(),
	                                                      outdegree,
	                                                      weight,
	                                                      syn_delay);
}

// 100% - 0
// 95% - 1
// 80% - 4
// 50% - 10
void init_extensor_flexor() {
	connect_fixed_outdegree(C3, inh_group3, 0.5, 15); // 0.5
	connect_fixed_outdegree(C4, inh_group4, 0.5, 15); // 0.5
	connect_fixed_outdegree(C5, inh_group5, 0.5, 15); // 0.5

	connect_fixed_outdegree(inh_group3, G1_3, 0.5, 20); // 20

	connect_fixed_outdegree(inh_group4, G1_3, 0.5, 20);
	connect_fixed_outdegree(inh_group4, G2_3, 0.5, 20);

	connect_fixed_outdegree(inh_group5, G1_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G2_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G3_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G4_3, 0.5, 20);

	/// D1
	// input from sensory
	connect_fixed_outdegree(C1, D1_1, 1, 0.4);
	connect_fixed_outdegree(C1, D1_4, 1, 0.4);
	connect_fixed_outdegree(C2, D1_1, 1, 0.4);
	connect_fixed_outdegree(C2, D1_4, 1, 0.4);
	// input from EES
	connect_fixed_outdegree(EES, D1_1, 2, 50); // ST value (?) // was 10
	connect_fixed_outdegree(EES, D1_4, 2, 20); // ST value (?) // was 10
	// inner connectomes
	connect_fixed_outdegree(D1_1, D1_2, 1, 1);
	connect_fixed_outdegree(D1_1, D1_3, 1, 10);
	connect_fixed_outdegree(D1_2, D1_1, 1, 7);
	connect_fixed_outdegree(D1_2, D1_3, 1, 13);
	connect_fixed_outdegree(D1_3, D1_1, 1, -30 * INH_COEF); // -10
	connect_fixed_outdegree(D1_3, D1_2, 1, -30 * INH_COEF); // -10
	connect_fixed_outdegree(D1_4, D1_3, 3, -30 * INH_COEF); // -20
	// output to
	connect_fixed_outdegree(D1_3, G1_1, 3, 6); // 8
	connect_fixed_outdegree(D1_3, ees_group1, 1.0, 60);

	// EES group connectomes
	connect_fixed_outdegree(ees_group1, ees_group2, 1, 20);

	/// D2
	// input from Sensory
	connect_fixed_outdegree(C2, D2_1, 1, 0.6); // was 8
	connect_fixed_outdegree(C2, D2_4, 1, 0.8);
	connect_fixed_outdegree(C3, D2_1, 1, 0.6); // was 8
	connect_fixed_outdegree(C3, D2_4, 1, 0.8);
	// input from Group (1)
	connect_fixed_outdegree(ees_group1, D2_1, 1.7, 0.8);
	connect_fixed_outdegree(ees_group1, D2_4, 1.7, 1);
	// inner connectomes
	connect_fixed_outdegree(D2_1, D2_2, 1, 3);
	connect_fixed_outdegree(D2_1, D2_3, 1, 10);
	connect_fixed_outdegree(D2_2, D2_1, 1, 7);
	connect_fixed_outdegree(D2_2, D2_3, 1, 20);
	connect_fixed_outdegree(D2_3, D2_1, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D2_3, D2_2, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D2_4, D2_3, 2, -20 * INH_COEF);
	// output to generator
	// connect_fixed_outdegree(D2_3, G2_1, 1.0, 8);	// FixMe moved

	// EES group connectomes
	connect_fixed_outdegree(ees_group2, ees_group3, 1, 20);

	/// D3
	// input from Sensory
	connect_fixed_outdegree(C3, D3_1, 1, 0.4); // was 0.5
	connect_fixed_outdegree(C3, D3_4, 1, 0.5);
	connect_fixed_outdegree(C4, D3_1, 1, 0.4); // was 0.5
	connect_fixed_outdegree(C4, D3_4, 1, 0.5);
	// input from Group (2)
	connect_fixed_outdegree(ees_group2, D3_1, 1, 1.0); // was 1.2
	connect_fixed_outdegree(ees_group2, D3_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D3_1, D3_2, 1, 3);
	connect_fixed_outdegree(D3_1, D3_3, 1, 10);
	connect_fixed_outdegree(D3_2, D3_1, 1, 7);
	connect_fixed_outdegree(D3_2, D3_3, 1, 20);
	connect_fixed_outdegree(D3_3, D3_1, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D3_3, D3_2, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D3_4, D3_3, 2, -10 * INH_COEF);
	// output to generator
	connect_fixed_outdegree(D3_3, G3_1, 1, 25);
	// suppression of the generator
	connect_fixed_outdegree(D3_3, G1_3, 1.5, 30);

	// EES group connectomes
	connect_fixed_outdegree(ees_group3, ees_group4, 2, 20);

	/// D4
	// input from Sensory
	connect_fixed_outdegree(C4, D4_1, 1, 0.4);
	connect_fixed_outdegree(C4, D4_4, 1, 0.5);
	connect_fixed_outdegree(C5, D4_1, 1, 0.4);
	connect_fixed_outdegree(C5, D4_4, 1, 0.5);
	// input from Group (3)
	connect_fixed_outdegree(ees_group3, D4_1, 1, 1.0);
	connect_fixed_outdegree(ees_group3, D4_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D4_1, D4_2, 1.0, 3);
	connect_fixed_outdegree(D4_1, D4_3, 1.0, 10);
	connect_fixed_outdegree(D4_2, D4_1, 1.0, 7);
	connect_fixed_outdegree(D4_2, D4_3, 1.0, 20);
	connect_fixed_outdegree(D4_3, D4_1, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_3, D4_2, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_4, D4_3, 2.0, -20 * INH_COEF);
	// output to the generator
	//	connect_fixed_outdegree(D4_3, G4_1, 3.0, 20.0);	//FixMe moved
	// suppression of the generator
	connect_fixed_outdegree(D4_3, G2_3, 1, 30);

	/// D5
	// input from Sensory
	connect_fixed_outdegree(C5, D5_1, 1, 0.5);
	connect_fixed_outdegree(C5, D5_4, 1, 0.5);
	// input from Group (4)
	connect_fixed_outdegree(ees_group4, D5_1, 1, 0.8); // was 1.1
	connect_fixed_outdegree(ees_group4, D5_4, 1, 1);
	// inner connectomes
	connect_fixed_outdegree(D5_1, D5_2, 1, 3);
	connect_fixed_outdegree(D5_1, D5_3, 1, 15);
	connect_fixed_outdegree(D5_2, D5_1, 1, 7);
	connect_fixed_outdegree(D5_2, D5_3, 1, 20);
	connect_fixed_outdegree(D5_3, D5_1, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D5_3, D5_2, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D5_4, D5_3, 2.5, -20 * INH_COEF);
	// output to the generator
	connect_fixed_outdegree(D5_3, G5_1, 3, 8);
	// suppression of the generators
	connect_fixed_outdegree(D5_3, G1_3, 1, 30);
	connect_fixed_outdegree(D5_3, G2_3, 1, 30);
	connect_fixed_outdegree(D5_3, G3_3, 1, 30);
	connect_fixed_outdegree(D5_3, G4_3, 1, 30);

	/// G1
	// inner connectomes
	connect_fixed_outdegree(G1_1, G1_2, 1, 10);
	connect_fixed_outdegree(G1_1, G1_3, 1, 15);
	connect_fixed_outdegree(G1_2, G1_1, 1, 10);
	connect_fixed_outdegree(G1_2, G1_3, 1, 15);
	connect_fixed_outdegree(G1_3, G1_1, 0.25, -20 * INH_COEF); // -70 - 40 // syn was 0.7
	connect_fixed_outdegree(G1_3, G1_2, 0.25, -20 * INH_COEF); // -70 - 40 // syn was 0.7
	// G1 -> IP_E
	connect_fixed_outdegree(G1_1, IP_E, 3, 20);
	connect_fixed_outdegree(G1_2, IP_E, 3, 20);
	// G1 -> IP_F
	connect_fixed_outdegree(G1_1, IP_F, 2.5, 15);
	connect_fixed_outdegree(G1_2, IP_F, 2.5, 15);
	/// G2
	// inner connectomes
	connect_fixed_outdegree(G2_1, G2_2, 1, 10);
	connect_fixed_outdegree(G2_1, G2_3, 1, 20);
	connect_fixed_outdegree(G2_2, G2_1, 1, 10);
	connect_fixed_outdegree(G2_2, G2_3, 1, 20);
	connect_fixed_outdegree(G2_3, G2_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G2_3, G2_2, 0.5, -30 * INH_COEF);
	// G2 -> IP_E
	connect_fixed_outdegree(G2_1, IP_E, 1, 20);
	connect_fixed_outdegree(G2_2, IP_E, 1, 20);
	// G2 -> IP_F
	connect_fixed_outdegree(G2_1, IP_F, 3, 20);
	connect_fixed_outdegree(G2_2, IP_F, 3, 20);

	/// G3
	// inner connectomes
	connect_fixed_outdegree(G3_1, G3_2, 1, 14);
	connect_fixed_outdegree(G3_1, G3_3, 1, 20);
	connect_fixed_outdegree(G3_2, G3_1, 1, 12);
	connect_fixed_outdegree(G3_2, G3_3, 1, 20);
	connect_fixed_outdegree(G3_3, G3_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G3_3, G3_2, 0.5, -30 * INH_COEF);
	// G3 -> IP_E
	connect_fixed_outdegree(G3_1, IP_E, 2, 25);
	connect_fixed_outdegree(G3_2, IP_E, 2, 25);
	// G3 -> IP_F
	connect_fixed_outdegree(G3_1, IP_F, 2.5, 20);
	connect_fixed_outdegree(G3_2, IP_F, 2.5, 20);

	/// G4
	// inner connectomes
	connect_fixed_outdegree(G4_1, G4_2, 1, 10);
	connect_fixed_outdegree(G4_1, G4_3, 1, 10);
	connect_fixed_outdegree(G4_2, G4_1, 1, 5);
	connect_fixed_outdegree(G4_2, G4_3, 1, 10);
	connect_fixed_outdegree(G4_3, G4_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G4_3, G4_2, 0.5, -30 * INH_COEF);
	// G4 -> IP_E
	connect_fixed_outdegree(G4_1, IP_E, 1, 17);
	connect_fixed_outdegree(G4_2, IP_E, 1, 17);
	// G4 -> IP_F
	connect_fixed_outdegree(G4_1, IP_F, 3, 17);
	connect_fixed_outdegree(G4_2, IP_F, 3, 17);

	/// G5
	// inner connectomes
	connect_fixed_outdegree(G5_1, G5_2, 1, 10);
	connect_fixed_outdegree(G5_1, G5_3, 1, 10);
	connect_fixed_outdegree(G5_2, G5_1, 1, 7);
	connect_fixed_outdegree(G5_2, G5_3, 1, 10);
	connect_fixed_outdegree(G5_3, G5_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G5_3, G5_2, 0.5, -30 * INH_COEF);
	// G5 -> IP_E
	connect_fixed_outdegree(G5_1, IP_E, 2, 20);
	connect_fixed_outdegree(G5_2, IP_E, 2, 20);
	// G5 -> IP_F
	connect_fixed_outdegree(G5_1, IP_F, 3, 20);
	connect_fixed_outdegree(G5_2, IP_F, 3, 20);
}

void init_ref_arc() {
	connect_fixed_outdegree(EES, Ia_Extensor, 1, 20); // was 20
	connect_fixed_outdegree(EES, Ia_Flexor, 1, 20); // was 20

	connect_fixed_outdegree(IP_E, MP_E, 1, 7); // 11 7
//	connect_fixed_outdegree(IP_E, Ia_E, 2.0, 20.0);
//
////	connect_fixed_outdegree(MP_E, Extensor, 2.0, 20.0);
//	connect_fixed_outdegree(MP_E, R_E, 2.0, 20.0);
//
	connect_fixed_outdegree(IP_F, MP_F, 1, 7); // 11 7
//	connect_fixed_outdegree(IP_F, Ia_F, 2.0, 20.0);
//
////	connect_fixed_outdegree(MP_F, Flexor, 2.0, 20.0);
//	connect_fixed_outdegree(MP_F, R_F, 2.0, 20.0);
//
//	connect_fixed_outdegree(Ib_F, Ib_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_F, MP_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_E, Ib_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_E, MP_E, 2.0, -5 * INH_COEF);
//
//	connect_fixed_outdegree(Ia_F, Ia_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ia_F, MP_E, 2.0, -5 * INH_COEF);
//	connect_fixed_outdegree(Ia_E, Ia_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ia_E, MP_F, 2.0, -20 * INH_COEF);
//
//	connect_fixed_outdegree(R_F, R_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_F, Ia_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_F, MP_F, 2.0, -20 * INH_COEF);
//
//	connect_fixed_outdegree(R_E, R_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_E, Ia_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_E, MP_E, 2.0, -5 * INH_COEF);

	connect_fixed_outdegree(Ia_Flexor, MP_F, 1, 10);
//	connect_fixed_outdegree(Ia, Ia_F, 1.0, 10.0);
//	connect_fixed_outdegree(Ia, Ib_F, 1.0, 10.0);

	connect_fixed_outdegree(Ia_Extensor, MP_E, 1, 10); // was 1 and 10
//	connect_fixed_outdegree(Ia, Ia_E, 1.0, 10.0);
//	connect_fixed_outdegree(Ia, Ib_E, 1.0, 10.0);
}

void save_result(int test_index,
                 float* voltage_recording,
                 float* current_recording,
                 int* spike_recording,
                 int neurons_number) {
	// save results for each neuron (voltage/current/spikes)
	char cwd[256];
	ofstream myfile;

	getcwd(cwd, sizeof(cwd));
	printf("[Test #%d] Save results to: %s \n", test_index, cwd);
	string new_name;


	new_name = "/volt_" + std::to_string(test_index) + ".dat";
	myfile.open(cwd + new_name);

	for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++){
		myfile << nrn_id << " ";
		for(int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++)
			myfile << voltage_recording[sim_iter + nrn_id * sim_time_in_step] << " ";
		myfile << "\n";
	}

	myfile.close();

//	new_name = "/curr_" + std::to_string(test_index) + ".dat";
//	myfile.open(cwd + new_name);
//
//	for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++){
//		myfile << nrn_id << " ";
//		for(int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++)
//			myfile << current_recording[sim_iter + nrn_id * sim_time_in_step] << " ";
//		myfile << "\n";
//	}
//
//	myfile.close();
//
//	new_name = "/spikes_" + std::to_string(test_index) + ".dat";
//	myfile.open(cwd + new_name);
//
//	for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++) {
//		myfile << nrn_id << " ";
//		for (int sim_iter = 0; sim_iter < sim_time_in_step; sim_iter++) {
//			float spike_time = spike_recording[sim_iter + nrn_id * sim_time_in_step] * sim_step;
//			if (spike_time != 0)
//				myfile << spike_time << " ";
//		}
//		myfile << "\n";
//	}
//
//	myfile.close();

}

template <typename type>
void memcpyHtD(type* gpu, type* host, int size) {
	hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice);
}

template <typename type>
void memcpyDtH(type* host, type* gpu, int size) {
	hipMemcpy(host, gpu, sizeof(type) * size, hipMemcpyDeviceToHost);
}

template <typename type>
unsigned int datasize(int size) {
	return sizeof(type) * size;
}

template <typename type>
void init_array(type *array, int size, type value){
	for(int i = 0; i < size; i++)
		array[i] = value;
}

__host__
void simulate(int test_index) {
	int neurons_number = static_cast<int>(metadatas.size());

	float* gpu_old_v;
	float* gpu_old_u;
	int* gpu_nrn_ref_time;
	int* gpu_nrn_ref_timer;
	bool* gpu_has_spike;
	float* gpu_nrn_current;
	int* gpu_synapses_number;

	// ToDo remove after debugging
	float* gpu_voltage_recording;
	float* gpu_current_recording;
	int* gpu_spike_recording;

	int synapses_number[neurons_number];

	float old_v[neurons_number];
	init_array<float>(old_v, neurons_number, V_rest);

	float old_u[neurons_number];
	init_array<float>(old_u, neurons_number, 0);

	int nrn_ref_time[neurons_number];
	init_array<int>(nrn_ref_time, neurons_number, ms_to_step(3.0));

	int nrn_ref_timer[neurons_number];
	init_array<int>(nrn_ref_timer, neurons_number, -1);

	bool has_spike[neurons_number];
	init_array<bool>(has_spike, neurons_number, false);

	float nrn_current[neurons_number];
	init_array<float>(nrn_current, neurons_number, 0);

	float multimeter_result[sim_time_in_step];
	init_array<float>(multimeter_result, sim_time_in_step, 0);

	// ToDo remove after debugging
	float* voltage_recording = (float *)malloc(datasize<float *>(neurons_number * sim_time_in_step));
	init_array<float>(voltage_recording, neurons_number * sim_time_in_step, 0);
	float* current_recording = (float *)malloc(datasize<float *>(neurons_number * sim_time_in_step));
	init_array<float>(current_recording, neurons_number * sim_time_in_step, 0);
	int* spike_recording = (int *)malloc(datasize<int *>(neurons_number * sim_time_in_step));
	init_array<int>(spike_recording, neurons_number * sim_time_in_step, 0);

	// init connectomes
	// REMOVE THEM AS [27:]
	/// connections which are inhibited by C=1
	// D1 -> G2
	// 1
	connect_fixed_outdegree(D1_3, G2_1, 0.5, 13);
	connect_fixed_outdegree(D1_3, inh_group5, sim_time_in_step, 0);	 // FixME FAKE
	// G5 -> I5
	// 1
	connect_fixed_outdegree(G5_1, inh_group5, 3, 20);
	connect_fixed_outdegree(G5_1, inh_group5, sim_time_in_step, 0);	 // FixME FAKE
	// 1
	connect_fixed_outdegree(G5_2, inh_group5, 3, 20);
	connect_fixed_outdegree(G5_2, inh_group5, sim_time_in_step, 0);	 // FixME FAKE
	// G3 -> G4
	// 1
	connect_fixed_outdegree(G3_1, G4_1, 1.0, 65);
	connect_fixed_outdegree(G3_1, inh_group5, sim_time_in_step, 0);	 // FixME FAKE
	// 1
	connect_fixed_outdegree(G3_2, G4_1, 1.0, 65);
	connect_fixed_outdegree(G3_2, inh_group5, sim_time_in_step, 0);	 // FixME FAKE
	// G2 -> D2
	// 2
	connect_fixed_outdegree(G2_1, D2_1, 1, 15);
	connect_fixed_outdegree(G2_1, D2_4, 1, 15);
	// 2
	connect_fixed_outdegree(G2_2, D2_1, 1, 15);
	connect_fixed_outdegree(G2_2, D2_4, 1, 15);
	// D2 -> D3
	// 2
	connect_fixed_outdegree(D2_3, D3_1, 0.5, 12.5);
	connect_fixed_outdegree(D2_3, D3_4, 0.5, 12.5);
	// G3 -> D4
	// 2
	connect_fixed_outdegree(G4_1, D4_1, 1, 65);
	connect_fixed_outdegree(G4_1, D4_4, 1, 65);
	// 2
	connect_fixed_outdegree(G4_2, D4_1, 1, 65);
	connect_fixed_outdegree(G4_2, D4_4, 1, 65);
	// D4 -> D5
	// 2
	connect_fixed_outdegree(D4_3, D5_1, 1, 10);
	connect_fixed_outdegree(D4_3, D5_4, 1, 10);

	/// end

	init_extensor_flexor();
	init_ref_arc();

	// REMOVE THEM AS [:-27]
	/// connections which are inhibited by C=0
	connect_fixed_outdegree(D2_3, G2_1, 1, 8);    // D2_3 IDs [0 ... 1]
	connect_fixed_outdegree(D4_3, G4_1, 3, 20);   // D4_3 IDs [20 ... 39]
	/// end

	int **gpu_synapses_post_nrn_id, **synapses_post_nrn_id = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay, **synapses_delay = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay_timer, **synapses_delay_timer = (int **)malloc(datasize<int* >(neurons_number));
	float **gpu_synapses_weight, **synapses_weight = (float **)malloc(datasize<float* >(neurons_number));

	// fill arrays of synapses
	for(int neuron_id = 0; neuron_id < neurons_number; neuron_id++) {
		int syn_count = static_cast<int>(metadatas.at(neuron_id).size());
		int tmp_synapses_post_nrn_id[syn_count];
		int tmp_synapses_delay[syn_count];
		int tmp_synapses_delay_timer[syn_count];
		float tmp_synapses_weight[syn_count];

		int syn_id = 0;
		for(SynapseMetadata metadata : metadatas.at(neuron_id)) {
			tmp_synapses_post_nrn_id[syn_id] = metadata.post_id;
			tmp_synapses_delay[syn_id] = metadata.synapse_delay;
			tmp_synapses_delay_timer[syn_id] = -1;
			tmp_synapses_weight[syn_id] = metadata.synapse_weight;
			syn_id++;
		}

		synapses_number[neuron_id] = syn_count;

		hipMalloc((void**)&synapses_post_nrn_id[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay_timer[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_weight[neuron_id], datasize<float>(syn_count));

		hipMemcpy(synapses_post_nrn_id[neuron_id], &tmp_synapses_post_nrn_id, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay[neuron_id], &tmp_synapses_delay, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay_timer[neuron_id], &tmp_synapses_delay_timer, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_weight[neuron_id], &tmp_synapses_weight, datasize<float>(syn_count), hipMemcpyHostToDevice);
	}

	hipMalloc((void ***)&gpu_synapses_post_nrn_id, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_post_nrn_id, synapses_post_nrn_id, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay, synapses_delay, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay_timer, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay_timer, synapses_delay_timer, neurons_number);

	hipMalloc((void ***)&gpu_synapses_weight, datasize<float *>(neurons_number));
	memcpyHtD<float *>(gpu_synapses_weight, synapses_weight, neurons_number);

	hipMalloc(&gpu_old_v, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_old_v, old_v, neurons_number);

	hipMalloc(&gpu_old_u, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_old_u, old_u, neurons_number);

	hipMalloc(&gpu_has_spike, datasize<bool>(neurons_number));
	memcpyHtD<bool>(gpu_has_spike, has_spike, neurons_number);

	hipMalloc(&gpu_nrn_ref_time, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_time, nrn_ref_time, neurons_number);

	hipMalloc(&gpu_nrn_ref_timer, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_timer, nrn_ref_timer, neurons_number);

	hipMalloc(&gpu_nrn_current, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_nrn_current, nrn_current, neurons_number);

	hipMalloc(&gpu_synapses_number, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_synapses_number, synapses_number, neurons_number);

	// FixMe debugging functionality
	hipMalloc(&gpu_voltage_recording, datasize<float>(neurons_number * sim_time_in_step));
	memcpyHtD<float>(gpu_voltage_recording, voltage_recording, neurons_number * sim_time_in_step);
	hipMalloc(&gpu_current_recording, datasize<float>(neurons_number * sim_time_in_step));
	memcpyHtD<float>(gpu_current_recording, current_recording, neurons_number * sim_time_in_step);
	hipMalloc(&gpu_spike_recording, datasize<int>(neurons_number * sim_time_in_step));
	memcpyHtD<int>(gpu_spike_recording, spike_recording, neurons_number * sim_time_in_step);

	int threads_per_block = 1024;
	int num_blocks = 1; // neurons_number / threads_per_block + 1;

	printf("Size of network: %i \n", neurons_number);
	printf("Start GPU with %d threads x %d blocks (Total: %d threads) \n",
	       threads_per_block, num_blocks, threads_per_block * num_blocks);

	// measure GPU ellapsed time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	sim_kernel<<<num_blocks, threads_per_block>>>(
	    gpu_old_v,
	    gpu_old_u,
	    gpu_nrn_current,
	    gpu_nrn_ref_time,
	    gpu_nrn_ref_timer,
	    gpu_synapses_number,
	    gpu_has_spike,
	    gpu_synapses_post_nrn_id,
	    gpu_synapses_delay,
	    gpu_synapses_delay_timer,
	    gpu_synapses_weight,
	    neurons_number,
	    // ToDo remove after debugging
	    gpu_voltage_recording,
	    gpu_current_recording,
	    gpu_spike_recording
	);

	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	double t = milliseconds / 1e3;
	double realtime_factor = T_sim / t / 1e3;
	printf("Ellapsed time: %fs. Realtime factor: x%f (%s than realtime)\n",
	       t, realtime_factor, realtime_factor > 1? "faster":"slower");

	// copy neurons/synapses array to the HOST
	// ToDo remove after debugging
	memcpyDtH<float>(voltage_recording, gpu_voltage_recording, neurons_number * sim_time_in_step);
	memcpyDtH<float>(current_recording, gpu_current_recording, neurons_number * sim_time_in_step);
	memcpyDtH<int>(spike_recording, gpu_spike_recording, neurons_number * sim_time_in_step);

	// tell the CPU to halt further processing until the CUDA kernel has finished doing its business
	hipDeviceSynchronize();

	save_result(test_index, voltage_recording, current_recording, spike_recording, neurons_number);

	hipDeviceReset();
}

int main(int argc, char* argv[]) {
	simulate(std::atoi(argv[1]));

	return 0;
}