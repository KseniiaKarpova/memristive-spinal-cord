#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <vector>
#include <ctime>
#include <cmath>
#include <stdexcept>
#include <random>
// for file writing
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <unistd.h>
// my classes
#include "Group.cpp"
// jetbrains cuda
#ifdef __JETBRAINS_IDE__
	#define __host__
	#define __device__
	#define __global__
	#define __shared__
#endif

/**
In this model, a spike is emitted if V_m >= V_T + 30 mV and V_m has fallen during the current time step

 6 cm/s = 125 [ms] has 30 slices
15 cm/s = 50 [ms] has 15 slices
21 cm/s = 25 [ms] has 6 slices

References:
  [1] https://en.wikipedia.org/wiki/Hodgkin–Huxley_model

**/

// parameters for variability of the simulation
const float INH_COEF = 1.0f;                  // strength coefficient of inhibitory synapses
const int EES_FREQ = 40;                      // [hz] spike frequency of EES
const int SENSORY_FREQ = 200;                 // [hz] spike frequency of C1-C5
const float T_SIMULATION = 10;                // [ms] simulation time
const float SIM_STEP = 0.025;                 // [s] simulation step
const int SPEED = 21;                         // [cm/s] speed of rat moving
const int skin_stim_time = 25;                // [ms] time of stimulating sensory (based on speed)
const int slices_number = 6;                  // number of slices (based on speed)

// stuff variables
unsigned int global_id = 0;                   // iter to count neurons one by one
const unsigned int syn_outdegree = 27;        // synapse number outgoing from one neuron
const unsigned int neurons_in_ip = 196;       // number of neurons in interneuronal pool
const unsigned int neurons_in_moto = 169;     // motoneurons number
const unsigned int neurons_in_group = 20;     // number of neurons in a group
const unsigned int neurons_in_afferent = 60;  // number of neurons in afferent

// neuron parameters
const float g_Na = 20000.0;  // [nS] Sodium peak conductance (Sodium voltage-gated ion channel)
const float g_K = 6000.0;    // [nS] Potassium peak conductance (Potassium voltage-gated ion channel)
const float g_L = 10.0;      // [nS] Leak conductance (Leak channels are represented by linear conductances)
const float C_m = 200.0;     // [pF] Capacity of the membrane (The lipid bilayer is represented as a capacitance)
// The electrochemical gradients driving the flow of ions are represented
// by voltage sources (E_X) whose voltages are determined by the ratio of the
// intra- and extracellular concentrations of the ionic species of interest [1]
const float E_Na = 50.0;     // [mV] Sodium reversal potential
const float E_K = -90.0f;    // [mV] Potassium reversal potential
const float E_L = -60.0f;    // [mV] Leak reversal potential
const float V_T = -63.0f;    // [mV] Voltage offset that controls dynamics. If V_T = -63mV => V_th = -50mV
const float E_ex = 0.0;      // [mV] Excitatory synaptic reversal potential
const float E_in = -80;      // [mV] Inhibitory synaptic reversal potential
const float tau_syn_exc = 5.0;  // [ms] Time constant of the excitatory synaptic exponential function
const float tau_syn_inh = 10.0; // [ms] Time constant of the inhibitory synaptic exponential function

// calculate spike frequency in steps [steps]
const unsigned int sensory_spike_each_step = (unsigned int)(1000 / SENSORY_FREQ / SIM_STEP);
const unsigned int ees_spike_each_step = (unsigned int)(1000 / EES_FREQ / SIM_STEP);
// calculate start time of CV spiking [steps]
const unsigned int CV1_begin_spiking_time = (unsigned int)(0.1 / SIM_STEP);
const unsigned int CV2_begin_spiking_time = (unsigned int)(skin_stim_time / SIM_STEP);
const unsigned int CV3_begin_spiking_time = (unsigned int)(2 * skin_stim_time / SIM_STEP);
const unsigned int CV4_begin_spiking_time = (unsigned int)(3 * skin_stim_time / SIM_STEP);
const unsigned int CV5_begin_spiking_time = (unsigned int)(5 * skin_stim_time / SIM_STEP);
// calculate end time of CV spiking [steps]
const unsigned int CV1_end_spiking_time = (unsigned int)(skin_stim_time / SIM_STEP);
const unsigned int CV2_end_spiking_time = (unsigned int)(2 * skin_stim_time / SIM_STEP);
const unsigned int CV3_end_spiking_time = (unsigned int)(3 * skin_stim_time / SIM_STEP);
const unsigned int CV4_end_spiking_time = (unsigned int)(5 * skin_stim_time / SIM_STEP);
const unsigned int CV5_end_spiking_time = (unsigned int)(6 * skin_stim_time / SIM_STEP);
// calculate steps activation of C0 and C1
const unsigned int steps_activation_C0 = (unsigned int)(skin_stim_time * 5 / SIM_STEP);
const unsigned int steps_activation_C1 = (unsigned int)(skin_stim_time * slices_number / SIM_STEP);
// calculate how much steps in simulation time [steps]
const unsigned int sim_time_in_steps = (unsigned int)(T_SIMULATION / SIM_STEP);

// struct for human-readable initialization of connectomes
struct SynapseMetadata {
	int post_id;
	int synapse_delay;
	float synapse_weight;

	SynapseMetadata() = default;
	SynapseMetadata(int post_id, float synapse_delay, float synapse_weight){
		this->post_id = post_id;
		this->synapse_delay = static_cast<int>(synapse_delay * (1 / SIM_STEP) + 0.5); // round
		this->synapse_weight = synapse_weight;
	}
};

// form structs of neurons global ID and groups name
Group form_group(string group_name, int nrns_in_group = neurons_in_group) {
	Group group = Group();
	group.group_name = group_name;
	group.id_start = global_id;
	group.id_end = global_id + nrns_in_group - 1;
	group.group_size = nrns_in_group;

	global_id += nrns_in_group;

	printf("Formed %s IDs [%d ... %d] = %d\n", group_name.c_str(), global_id - nrns_in_group, global_id - 1, nrns_in_group);

	return group;
}

/* Nodes with changable connectomes
C=1 :: disable for neurons 0 <= tid <= 100 and their first 27 synapses. Slice as [54:]
[D2_3, D4_3, D1_3, G2_1, G2_2, G3_1, G3_2, G4_1, G4_2, G5_1, G5_2] */

// Form neuron groups
// At first init nodes with changable connectomes to reduce "and" operators at synapse checking (by tid)

// inhibited by C=1 group
Group D1_3 = form_group("D1_3");	// D1_3 IDs [0 ... 19]
Group D2_3 = form_group("D2_3");
Group D4_3 = form_group("D4_3");
Group G3_1 = form_group("G3_1");
Group G3_2 = form_group("G3_2");	// G3_2 IDs [80 ... 99]

// groups of neurons with generators
Group CV1 = form_group("CV1");
Group CV2 = form_group("CV2");
Group CV3 = form_group("CV3");
Group CV4 = form_group("CV4");
Group CV5 = form_group("CV5");
Group EES = form_group("EES");

// groups of neurons without changable synapses
Group D1_1 = form_group("D1_1");
Group D1_2 = form_group("D1_2");
// Group D1_3 inited in the group above
Group D1_4 = form_group("D1_4");

Group D2_1 = form_group("D2_1");
Group D2_2 = form_group("D2_2");
// Group D2_3 inited in the group above
Group D2_4 = form_group("D2_4");

Group D3_1 = form_group("D3_1");
Group D3_2 = form_group("D3_2");
Group D3_3 = form_group("D3_3");
Group D3_4 = form_group("D3_4");

Group D4_1 = form_group("D4_1");
Group D4_2 = form_group("D4_2");
// Group D4_3 inited in the group above
Group D4_4 = form_group("D4_4");

Group D5_1 = form_group("D5_1");
Group D5_2 = form_group("D5_2");
Group D5_3 = form_group("D5_3");
Group D5_4 = form_group("D5_4");

Group G1_1 = form_group("G1_1");
Group G1_2 = form_group("G1_2");
Group G1_3 = form_group("G1_3");

Group G2_1 = form_group("G2_1");
Group G2_2 = form_group("G2_2");
Group G2_3 = form_group("G2_3");

// Group G3_1 inited in the group above
// Group G3_2 inited in the group above
Group G3_3 = form_group("G3_3");

Group G4_1 = form_group("G4_1");
Group G4_2 = form_group("G4_2");
Group G4_3 = form_group("G4_3");

Group G5_1 = form_group("G5_1");
Group G5_2 = form_group("G5_2");
Group G5_3 = form_group("G5_3");

Group IP_E = form_group("IP_E", neurons_in_ip);
Group IP_F = form_group("IP_F", neurons_in_ip);

Group MP_E = form_group("MP_E", neurons_in_moto);
Group MP_F = form_group("MP_F", neurons_in_moto);

Group Ia_Extensor = form_group("Ia_Extensor", neurons_in_afferent);
Group Ia_Flexor = form_group("Ia_Flexor", neurons_in_afferent);

Group inh_group3 = form_group("inh_group3");
Group inh_group4 = form_group("inh_group4");
Group inh_group5 = form_group("inh_group5");

Group ees_group1 = form_group("ees_group1");
Group ees_group2 = form_group("ees_group2");
Group ees_group3 = form_group("ees_group3");
Group ees_group4 = form_group("ees_group4");

Group R_E = form_group("R_E");
Group R_F = form_group("R_F");

Group Ia_E = form_group("Ia_E");
Group Ia_F = form_group("Ia_F");
Group Ib_E = form_group("Ib_E");
Group Ib_F = form_group("Ib_F");

// global vectors of SynapseMetadata of synapses for each neuron
vector<vector<SynapseMetadata>> metadatas(global_id, vector<SynapseMetadata>());

__host__
int ms_to_step(float ms) { return (int)(ms / SIM_STEP); }

__global__
void GPU_simulation(float *V_m,
					float *h,
					float *m,
					float *n,
					float *g_exc,
					float *g_inh,
					bool *has_spike,
					int *nrn_ref_time,
					int *nrn_ref_time_timer,
					int *synapses_number,
					int **synapses_post_nrn_id,
					int **synapses_delay,
					int **synapses_delay_timer,
					float **synapses_weight,
					unsigned int sim_iter,
					int activated_C_,
					int shift_time_by_step) {
	// activated_C_ 0 - at flexor (TA)
	// activated_C_ 1 - at extensor (MG)

	__shared__ short decrease_lvl_Ia_spikes; // level of inhibition. 3 - no inh., 2 - small, 1 - strong
	__shared__ bool sensory_spike_flag;      // flag which denotes is time for spiking

	// get id of the GPU thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// reset spike flag of the current neuron before calculations
	has_spike[tid] = false;



	if (tid == 0) {
		decrease_lvl_Ia_spikes = 1;
		sensory_spike_flag = sim_iter % sensory_spike_each_step == 0;
	}

	// wait all threads
	__syncthreads();

	// activate only sensory C1-CV5 and control Ia afferent spikes
	if (100 <= tid && tid <= 199) {
		int shifted_sim_iter = sim_iter - shift_time_by_step;
		// CV1
		if (100 <= tid && tid <= 119 && (CV1_begin_spiking_time <= shifted_sim_iter) &&
			(shifted_sim_iter < CV1_end_spiking_time)) {
			if (tid == 100) {
				decrease_lvl_Ia_spikes = 3;
			}
			if (activated_C_ == 1 && sensory_spike_flag) {
				g_exc[tid] = 5000; // set spike state
			}
		} else {
			// CV2
			if (120 <= tid && tid <= 139 && (CV2_begin_spiking_time <= shifted_sim_iter) &&
				(shifted_sim_iter < CV2_end_spiking_time)) {
				if (tid == 120) {
					decrease_lvl_Ia_spikes = 2;
				}
				if (activated_C_ == 1 && sensory_spike_flag) {
					g_exc[tid] = 5000; // set spike state
				}
			} else {
				// CV3
				if (140 <= tid && tid <= 159 && (CV3_begin_spiking_time <= shifted_sim_iter) &&
					(shifted_sim_iter < CV3_end_spiking_time)) {
					if (tid == 140) {
						decrease_lvl_Ia_spikes = 1;
					}
					if (activated_C_ == 1 && sensory_spike_flag) {
						g_exc[tid] = 5000; // set spike state
					}
				} else {
					// CV4
					if (160 <= tid && tid <= 179 && (CV4_begin_spiking_time <= shifted_sim_iter) &&
						(shifted_sim_iter < CV4_end_spiking_time)) {
						if (tid == 160) {
							decrease_lvl_Ia_spikes = 2;
						}
						if (activated_C_ == 1 && sensory_spike_flag) {
							g_exc[tid] = 5000; // set spike state
						}
					} else {
						// CV5
						if (180 <= tid && tid <= 199 && (CV5_begin_spiking_time <= shifted_sim_iter) &&
							(shifted_sim_iter < CV5_end_spiking_time)) {
							if (tid == 180) {
								decrease_lvl_Ia_spikes = 3;
							}
							if (activated_C_ == 1 && sensory_spike_flag) {
								g_exc[tid] = 5000; // set spike state
							}
						}
					}
				}
			}
		}
	}

	__syncthreads();

	/// NEURONS UPDATING

	// generating spikes for EES
	if (200 <= tid && tid <= 219 && (sim_iter % ees_spike_each_step == 0)) {
		g_exc[tid] = 5000;
	}

	// Ia IDs [1550 ... 1669], control spike number of Ia afferent by resetting neuron current
	if (1550 <= tid && tid <= 1669) {
		// rule for the 2nd level
		if (decrease_lvl_Ia_spikes == 2 && tid % 3 == 0) {
			// reset current of 1/3 of neurons
			g_exc[tid] = 0;
		} else {
			// rule for the 3rd level
			if (decrease_lvl_Ia_spikes == 3 && tid % 2 == 0) {
				// reset current of 1/2 of neurons
				g_exc[tid] = 0;
			}
		}
	}

	// inhibit IP_E IDs [820 ... 1015] by C0
	if (activated_C_ == 0 && 820 <= tid && tid <= 1015) {
		g_exc[tid] *= (1 - INH_COEF);
	}
	// inhibit IP_F IDs [1016 ... 1211] by C1
	if (activated_C_ == 1 && 1016 <= tid && tid <= 1211) {
		g_exc[tid] *= (1 - INH_COEF);
	}
	// inhibit Ia_Extensor IDs [1550 ... 1609] by C0
	if (activated_C_ == 0 && 1550 <= tid && tid <= 1609) {
		g_exc[tid] *= (1 - INH_COEF);
	}
	// inhibit Ia_Flexor IDs [1610 ... 1669] by C1
	if (activated_C_ == 1 && 1610 <= tid && tid <= 1669) {
		g_exc[tid] *= (1 - INH_COEF);
	}

	float V_m_old = V_m[tid];

	// ToDo check this with biological data (aprx)
	// the maximal value of input current (10 000 pA = 10 nA)
	if (g_exc[tid] > 10000)
		g_exc[tid] = 10000;
//	if (g_exc[tid] < -10000)
//		g_exc[tid] = -10000;

	// ionic currents
	const float I_Na = g_Na * std::pow(m[tid], 3) * h[tid] * (V_m_old - E_Na);
	const float I_K = g_K * std::pow(n[tid], 4) * (V_m_old - E_K);
	const float I_L = g_L * (V_m_old - E_L);

	const float I_syn_exc = g_exc[tid] * (V_m_old - E_ex);
	const float I_syn_inh = g_inh[tid] * (V_m_old - E_in);

	if (nrn_ref_time_timer[tid] > 0) {
		// if neuron in the refractory period
		// calculate V without I syn
		// membrane potential
		V_m[tid] = (-I_Na - I_K - I_L) / C_m;
	} else {
		// membrane potential
		V_m[tid] = (-I_Na - I_K - I_L - I_syn_exc - I_syn_inh) / C_m;
	}

	// channel dynamics
	const float V = V_m[tid] - V_T;

	// alpha_X and beta_X are rate constants for the X ion channel, which depend on voltage but not time
	const float alpha_n = 0.032 * (15.0 - V) / (std::exp((15.0 - V) / 5.0) - 1.0);
	const float beta_n = 0.5 * std::exp((10.0 - V) / 40);
	const float alpha_m = 0.32 * (13.0 - V) / (std::exp((13.0 - V) / 4.0) - 1.0);
	const float beta_m = 0.28 * (V - 40.0) / (std::exp((V - 40.0) / 5.0) - 1.0);
	const float alpha_h = 0.128 * std::exp((17.0 - V) / 18.0);
	const float beta_h = 4.0 / (1.0 + std::exp((40.0 - V) / 5.0));

	// n, m, and h are dimensionless quantities between 0 and 1 that are associated with
	// potassium channel activation, sodium channel activation, and sodium channel inactivation, respectively
	m[tid] = alpha_m - (alpha_m + beta_m) * m[tid];
	h[tid] = alpha_h - (alpha_h + beta_h) * h[tid];
	n[tid] = alpha_n - (alpha_n + beta_n) * n[tid];

	// synapses: exponential conductance
	g_exc[tid] = -g_exc[tid] / tau_syn_exc;
	g_inh[tid] = -g_inh[tid] / tau_syn_inh;

	// (threshold && maximal peak)
	if (V_m[tid] >= V_T + 30.0 && V_m_old > V_m[tid]) {
		has_spike[tid] = true;
		// set the refractory period
		nrn_ref_time_timer[tid] = nrn_ref_time[tid];
	}

	__syncthreads();

	/// SYNAPSE UPDATING

	// init basic synapse ids
	int syn_id_begin = 0;
	int syn_id_end = synapses_number[tid];

	// C=1 -- "slice" as [54:] -- skip the first 54 synapses because they must be inhibited
	if (activated_C_ == 1 && 0 <= tid && tid <= 99) {
		syn_id_begin = 54; // 27 * 2
	}

	// pointers to current neuronID synapses_delay_timer (decrease array calls)
	int *ptr_delay_timers = synapses_delay_timer[tid];

	// synapse updating loop (with formed begin/end borders)
	for (int syn_id = syn_id_begin; syn_id < syn_id_end; syn_id++) {
		// add synaptic delay if neuron has spike
		if (has_spike[tid] && ptr_delay_timers[syn_id] == -1) {
			ptr_delay_timers[syn_id] = synapses_delay[tid][syn_id];
		}
		// if synaptic delay is zero it means the time when synapse increase I by synaptic weight
		if (ptr_delay_timers[syn_id] == 0) {
			// post neuron ID = synapses_post_nrn_id[tid][syn_id], thread-safe (!)
			if (synapses_weight[tid][syn_id] >= 0) {
				atomicAdd(&g_exc[synapses_post_nrn_id[tid][syn_id]], synapses_weight[tid][syn_id]);
			} else {
				atomicAdd(&g_inh[synapses_post_nrn_id[tid][syn_id]], synapses_weight[tid][syn_id]);
			}
			// make synapse timer a "free" for next spikes
			ptr_delay_timers[syn_id] = -1;
		}
		// update synapse delay timer
		if (ptr_delay_timers[syn_id] > 0) {
			ptr_delay_timers[syn_id]--;
		}
	} // end synapse updating loop

	// update the refractory period timer
	if (nrn_ref_time_timer[tid] > 0)
		nrn_ref_time_timer[tid]--;

} // end of GPU kernel


void connect_fixed_outdegree(Group pre_neurons, Group post_neurons,
                             float syn_delay, float weight, int outdegree = syn_outdegree) {
	// connect neurons with uniform distribution and normal distributon for syn delay and weight
	weight *= (100 * 0.7);

	random_device rd;
	mt19937 gen(rd());	// Initialize pseudo-random number generator

	uniform_int_distribution<int> id_distr(post_neurons.id_start, post_neurons.id_end);
	normal_distribution<float> delay_distr(syn_delay, syn_delay / 10);
	normal_distribution<float> weight_distr(weight, weight / 10);

	#ifdef DEBUG
	printf("pre group %s (%d, %d) to post %s (%d, %d)\n",
	       pre_neurons.group_name.c_str(),
	       pre_neurons.id_start,
	       pre_neurons.id_end,
	       post_neurons.group_name.c_str(),
	       post_neurons.id_start,
	       post_neurons.id_end);
	#endif

	for (int pre_id = pre_neurons.id_start; pre_id <= pre_neurons.id_end; pre_id++) {
		for (int i = 0; i < outdegree; i++) {
			int rand_post_id = id_distr(gen);
			float syn_delay_dist = syn_delay; //delay_distr(gen);
			float syn_weight_dist = weight; //weight_distr(gen);
			#ifdef DEBUG
			printf("weight %f (%f), delay %f (%f) \n",
					syn_weight_dist, weight, syn_delay_dist, syn_delay);
			#endif
			metadatas.at(pre_id).push_back(SynapseMetadata(rand_post_id, syn_delay_dist, syn_weight_dist));
		}
	}

	printf("Connect %s with %s (1:%d). W=%.2f, D=%.1f\n", pre_neurons.group_name.c_str(),
	                                                      post_neurons.group_name.c_str(),
	                                                      outdegree,
	                                                      weight,
	                                                      syn_delay);
}


void init_extensor_flexor() {
	connect_fixed_outdegree(CV3, inh_group3, 0.5, 15); // 0.5
	connect_fixed_outdegree(CV4, inh_group4, 0.5, 15); // 0.5
	connect_fixed_outdegree(CV5, inh_group5, 0.5, 15); // 0.5

	connect_fixed_outdegree(inh_group3, G1_3, 0.5, 20); // 20

	connect_fixed_outdegree(inh_group4, G1_3, 0.5, 20);
	connect_fixed_outdegree(inh_group4, G2_3, 0.5, 20);

	connect_fixed_outdegree(inh_group5, G1_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G2_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G3_3, 0.5, 20);
	connect_fixed_outdegree(inh_group5, G4_3, 0.5, 20);

	/// D1
	// input from sensory
	connect_fixed_outdegree(CV1, D1_1, 1, 0.4);
	connect_fixed_outdegree(CV1, D1_4, 1, 0.4);
	connect_fixed_outdegree(CV2, D1_1, 1, 0.4);
	connect_fixed_outdegree(CV2, D1_4, 1, 0.4);
	// input from EES
	connect_fixed_outdegree(EES, D1_1, 2, 50); // ST value (?) // was 10
	connect_fixed_outdegree(EES, D1_4, 2, 20); // ST value (?) // was 10
	// inner connectomes
	connect_fixed_outdegree(D1_1, D1_2, 1, 1);
	connect_fixed_outdegree(D1_1, D1_3, 1, 10);
	connect_fixed_outdegree(D1_2, D1_1, 1, 7);
	connect_fixed_outdegree(D1_2, D1_3, 1, 13);
	connect_fixed_outdegree(D1_3, D1_1, 1, -30 * INH_COEF); // -10
	connect_fixed_outdegree(D1_3, D1_2, 1, -30 * INH_COEF); // -10
	connect_fixed_outdegree(D1_4, D1_3, 3, -30 * INH_COEF); // -20
	// output to
	connect_fixed_outdegree(D1_3, G1_1, 3, 6); // 8
	connect_fixed_outdegree(D1_3, ees_group1, 1.0, 60);

	// EES group connectomes
	connect_fixed_outdegree(ees_group1, ees_group2, 1, 20);

	/// D2
	// input from Sensory
	connect_fixed_outdegree(CV2, D2_1, 1, 0.6); // was 8
	connect_fixed_outdegree(CV2, D2_4, 1, 0.8);
	connect_fixed_outdegree(CV3, D2_1, 1, 0.6); // was 8
	connect_fixed_outdegree(CV3, D2_4, 1, 0.8);
	// input from Group (1)
	connect_fixed_outdegree(ees_group1, D2_1, 1.7, 0.8);
	connect_fixed_outdegree(ees_group1, D2_4, 1.7, 1);
	// inner connectomes
	connect_fixed_outdegree(D2_1, D2_2, 1, 3);
	connect_fixed_outdegree(D2_1, D2_3, 1, 10);
	connect_fixed_outdegree(D2_2, D2_1, 1, 7);
	connect_fixed_outdegree(D2_2, D2_3, 1, 20);
	connect_fixed_outdegree(D2_3, D2_1, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D2_3, D2_2, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D2_4, D2_3, 2, -20 * INH_COEF);
	// output to generator
	connect_fixed_outdegree(D2_3, G2_1, 1, 8);

	// EES group connectomes
	connect_fixed_outdegree(ees_group2, ees_group3, 1, 20);

	/// D3
	// input from Sensory
	connect_fixed_outdegree(CV3, D3_1, 1, 0.4); // was 0.5
	connect_fixed_outdegree(CV3, D3_4, 1, 0.5);
	connect_fixed_outdegree(CV4, D3_1, 1, 0.4); // was 0.5
	connect_fixed_outdegree(CV4, D3_4, 1, 0.5);
	// input from Group (2)
	connect_fixed_outdegree(ees_group2, D3_1, 1, 1.0); // was 1.2
	connect_fixed_outdegree(ees_group2, D3_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D3_1, D3_2, 1, 3);
	connect_fixed_outdegree(D3_1, D3_3, 1, 10);
	connect_fixed_outdegree(D3_2, D3_1, 1, 7);
	connect_fixed_outdegree(D3_2, D3_3, 1, 20);
	connect_fixed_outdegree(D3_3, D3_1, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D3_3, D3_2, 1, -10 * INH_COEF);
	connect_fixed_outdegree(D3_4, D3_3, 2, -10 * INH_COEF);
	// output to generator
	connect_fixed_outdegree(D3_3, G3_1, 1, 25);
	// suppression of the generator
	connect_fixed_outdegree(D3_3, G1_3, 1.5, 30);

	// EES group connectomes
	connect_fixed_outdegree(ees_group3, ees_group4, 2, 20);

	/// D4
	// input from Sensory
	connect_fixed_outdegree(CV4, D4_1, 1, 0.4);
	connect_fixed_outdegree(CV4, D4_4, 1, 0.5);
	connect_fixed_outdegree(CV5, D4_1, 1, 0.4);
	connect_fixed_outdegree(CV5, D4_4, 1, 0.5);
	// input from Group (3)
	connect_fixed_outdegree(ees_group3, D4_1, 1, 1.0);
	connect_fixed_outdegree(ees_group3, D4_4, 1, 1.2);
	// inner connectomes
	connect_fixed_outdegree(D4_1, D4_2, 1.0, 3);
	connect_fixed_outdegree(D4_1, D4_3, 1.0, 10);
	connect_fixed_outdegree(D4_2, D4_1, 1.0, 7);
	connect_fixed_outdegree(D4_2, D4_3, 1.0, 20);
	connect_fixed_outdegree(D4_3, D4_1, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_3, D4_2, 1.0, -20 * INH_COEF);
	connect_fixed_outdegree(D4_4, D4_3, 2.0, -20 * INH_COEF);
	// output to the generator
	connect_fixed_outdegree(D4_3, G4_1, 3, 20);
	// suppression of the generator
	connect_fixed_outdegree(D4_3, G2_3, 1, 30);

	/// D5
	// input from Sensory
	connect_fixed_outdegree(CV5, D5_1, 1, 0.5);
	connect_fixed_outdegree(CV5, D5_4, 1, 0.5);
	// input from Group (4)
	connect_fixed_outdegree(ees_group4, D5_1, 1, 0.8); // was 1.1
	connect_fixed_outdegree(ees_group4, D5_4, 1, 1);
	// inner connectomes
	connect_fixed_outdegree(D5_1, D5_2, 1, 3);
	connect_fixed_outdegree(D5_1, D5_3, 1, 15);
	connect_fixed_outdegree(D5_2, D5_1, 1, 7);
	connect_fixed_outdegree(D5_2, D5_3, 1, 20);
	connect_fixed_outdegree(D5_3, D5_1, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D5_3, D5_2, 1, -20 * INH_COEF);
	connect_fixed_outdegree(D5_4, D5_3, 2.5, -20 * INH_COEF);
	// output to the generator
	connect_fixed_outdegree(D5_3, G5_1, 3, 8);
	// suppression of the generators
	connect_fixed_outdegree(D5_3, G1_3, 1, 30);
	connect_fixed_outdegree(D5_3, G2_3, 1, 30);
	connect_fixed_outdegree(D5_3, G3_3, 1, 30);
	connect_fixed_outdegree(D5_3, G4_3, 1, 30);

	/// G1
	// inner connectomes
	connect_fixed_outdegree(G1_1, G1_2, 1, 10);
	connect_fixed_outdegree(G1_1, G1_3, 1, 15);
	connect_fixed_outdegree(G1_2, G1_1, 1, 10);
	connect_fixed_outdegree(G1_2, G1_3, 1, 15);
	connect_fixed_outdegree(G1_3, G1_1, 0.25, -20 * INH_COEF); // -70 - 40 // syn was 0.7
	connect_fixed_outdegree(G1_3, G1_2, 0.25, -20 * INH_COEF); // -70 - 40 // syn was 0.7
	// G1 -> IP_E
	connect_fixed_outdegree(G1_1, IP_E, 3, 20);
	connect_fixed_outdegree(G1_2, IP_E, 3, 20);
	// G1 -> IP_F
	connect_fixed_outdegree(G1_1, IP_F, 2.5, 15);
	connect_fixed_outdegree(G1_2, IP_F, 2.5, 15);
	/// G2
	// inner connectomes
	connect_fixed_outdegree(G2_1, G2_2, 1, 10);
	connect_fixed_outdegree(G2_1, G2_3, 1, 20);
	connect_fixed_outdegree(G2_2, G2_1, 1, 10);
	connect_fixed_outdegree(G2_2, G2_3, 1, 20);
	connect_fixed_outdegree(G2_3, G2_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G2_3, G2_2, 0.5, -30 * INH_COEF);
	// G2 -> IP_E
	connect_fixed_outdegree(G2_1, IP_E, 1, 20);
	connect_fixed_outdegree(G2_2, IP_E, 1, 20);
	// G2 -> IP_F
	connect_fixed_outdegree(G2_1, IP_F, 3, 20);
	connect_fixed_outdegree(G2_2, IP_F, 3, 20);

	/// G3
	// inner connectomes
	connect_fixed_outdegree(G3_1, G3_2, 1, 14);
	connect_fixed_outdegree(G3_1, G3_3, 1, 20);
	connect_fixed_outdegree(G3_2, G3_1, 1, 12);
	connect_fixed_outdegree(G3_2, G3_3, 1, 20);
	connect_fixed_outdegree(G3_3, G3_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G3_3, G3_2, 0.5, -30 * INH_COEF);
	// G3 -> IP_E
	connect_fixed_outdegree(G3_1, IP_E, 2, 25);
	connect_fixed_outdegree(G3_2, IP_E, 2, 25);
	// G3 -> IP_F
	connect_fixed_outdegree(G3_1, IP_F, 2.5, 20);
	connect_fixed_outdegree(G3_2, IP_F, 2.5, 20);

	/// G4
	// inner connectomes
	connect_fixed_outdegree(G4_1, G4_2, 1, 10);
	connect_fixed_outdegree(G4_1, G4_3, 1, 10);
	connect_fixed_outdegree(G4_2, G4_1, 1, 5);
	connect_fixed_outdegree(G4_2, G4_3, 1, 10);
	connect_fixed_outdegree(G4_3, G4_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G4_3, G4_2, 0.5, -30 * INH_COEF);
	// G4 -> IP_E
	connect_fixed_outdegree(G4_1, IP_E, 1, 17);
	connect_fixed_outdegree(G4_2, IP_E, 1, 17);
	// G4 -> IP_F
	connect_fixed_outdegree(G4_1, IP_F, 3, 17);
	connect_fixed_outdegree(G4_2, IP_F, 3, 17);

	/// G5
	// inner connectomes
	connect_fixed_outdegree(G5_1, G5_2, 1, 10);
	connect_fixed_outdegree(G5_1, G5_3, 1, 10);
	connect_fixed_outdegree(G5_2, G5_1, 1, 7);
	connect_fixed_outdegree(G5_2, G5_3, 1, 10);
	connect_fixed_outdegree(G5_3, G5_1, 0.5, -30 * INH_COEF);
	connect_fixed_outdegree(G5_3, G5_2, 0.5, -30 * INH_COEF);
	// G5 -> IP_E
	connect_fixed_outdegree(G5_1, IP_E, 2, 20);
	connect_fixed_outdegree(G5_2, IP_E, 2, 20);
	// G5 -> IP_F
	connect_fixed_outdegree(G5_1, IP_F, 3, 20);
	connect_fixed_outdegree(G5_2, IP_F, 3, 20);
}

void init_ref_arc() {
	connect_fixed_outdegree(EES, Ia_Extensor, 1, 20); // was 20
	connect_fixed_outdegree(EES, Ia_Flexor, 1, 20); // was 20

	connect_fixed_outdegree(IP_E, MP_E, 1, 7); // 11 7
//	connect_fixed_outdegree(IP_E, Ia_E, 2.0, 20.0);
//
////	connect_fixed_outdegree(MP_E, Extensor, 2.0, 20.0);
//	connect_fixed_outdegree(MP_E, R_E, 2.0, 20.0);
//
	connect_fixed_outdegree(IP_F, MP_F, 1, 7); // 11 7
//	connect_fixed_outdegree(IP_F, Ia_F, 2.0, 20.0);
//
////	connect_fixed_outdegree(MP_F, Flexor, 2.0, 20.0);
//	connect_fixed_outdegree(MP_F, R_F, 2.0, 20.0);
//
//	connect_fixed_outdegree(Ib_F, Ib_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_F, MP_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_E, Ib_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ib_E, MP_E, 2.0, -5 * INH_COEF);
//
//	connect_fixed_outdegree(Ia_F, Ia_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ia_F, MP_E, 2.0, -5 * INH_COEF);
//	connect_fixed_outdegree(Ia_E, Ia_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(Ia_E, MP_F, 2.0, -20 * INH_COEF);
//
//	connect_fixed_outdegree(R_F, R_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_F, Ia_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_F, MP_F, 2.0, -20 * INH_COEF);
//
//	connect_fixed_outdegree(R_E, R_F, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_E, Ia_E, 2.0, -20 * INH_COEF);
//	connect_fixed_outdegree(R_E, MP_E, 2.0, -5 * INH_COEF);

	connect_fixed_outdegree(Ia_Flexor, MP_F, 1, 10);
//	connect_fixed_outdegree(Ia, Ia_F, 1.0, 10.0);
//	connect_fixed_outdegree(Ia, Ib_F, 1.0, 10.0);

	connect_fixed_outdegree(Ia_Extensor, MP_E, 1, 10); // was 1 and 10
//	connect_fixed_outdegree(Ia, Ia_E, 1.0, 10.0);
//	connect_fixed_outdegree(Ia, Ib_E, 1.0, 10.0);
}

void save_result(int test_index,
                 float* voltage_recording,
                 float* current_recording,
                 int* spike_recording,
                 int neurons_number, int full_save) {
	// save results for each neuron (voltage/current/spikes)
	char cwd[256];
	ofstream myfile;

	getcwd(cwd, sizeof(cwd));
	printf("[Test #%d] Save results to: %s \n", test_index, cwd);
	string new_name;


	new_name = "/volt_" + std::to_string(test_index) + ".dat";
	myfile.open(cwd + new_name);

	for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++){
		myfile << nrn_id << " ";
		for(int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++)
			myfile << voltage_recording[sim_iter + nrn_id * sim_time_in_steps] << " ";
		myfile << "\n";
	}

	myfile.close();

	if (full_save == 1) {
		new_name = "/curr_" + std::to_string(test_index) + ".dat";
		myfile.open(cwd + new_name);

		for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++){
			myfile << nrn_id << " ";
			for(int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++)
				myfile << current_recording[sim_iter + nrn_id * sim_time_in_steps] << " ";
			myfile << "\n";
		}

		myfile.close();

		new_name = "/spikes_" + std::to_string(test_index) + ".dat";
		myfile.open(cwd + new_name);

		for(int nrn_id = 0; nrn_id < neurons_number; nrn_id++) {
			myfile << nrn_id << " ";
			for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++) {
				float spike_time = spike_recording[sim_iter + nrn_id * sim_time_in_steps] * SIM_STEP;
				if (spike_time != 0)
					myfile << spike_time << " ";
			}
			myfile << "\n";
		}

		myfile.close();
	}
}

template <typename type>
void memcpyHtD(type* gpu, type* host, int size) {
	hipMemcpy(gpu, host, sizeof(type) * size, hipMemcpyHostToDevice);
}

template <typename type>
void memcpyDtH(type* host, type* gpu, int size) {
	hipMemcpy(host, gpu, sizeof(type) * size, hipMemcpyDeviceToHost);
}

template <typename type>
unsigned int datasize(int size) {
	return sizeof(type) * size;
}

template <typename type>
void init_array(type *array, int size, type value){
	for(int i = 0; i < size; i++)
		array[i] = value;
}

__host__
void simulate(int test_index, int full_save) {
	int neurons_number = static_cast<int>(metadatas.size());

	/// init values
	const float alpha_n = 0.032 * (15 - E_L) / (std::exp((15 - E_L) / 5) - 1);
	const float beta_n = 0.5 * std::exp((10 - E_L) / 40);
	const float alpha_m = 0.32 * (13 - E_L) / (std::exp((13 - E_L) / 4) - 1);
	const float beta_m = 0.28 * (E_L - 40) / (std::exp((E_L - 40) / 5) - 1);
	const float alpha_h = 0.128 * std::exp((17. - E_L) / 18);
	const float beta_h = 4 / (1 + std::exp((40 - E_L) / 5));

	/// neurons parameters
	float* gpu_v_m;
	int* gpu_nrn_ref_time;
	int* gpu_nrn_ref_time_timer;
	bool* gpu_has_spike;
	int* gpu_synapses_number;
	float* gpu_n;
	float* gpu_h;
	float* gpu_m;
	float* gpu_g_exc;
	float* gpu_g_inh;

	// neuron membrane potential
	float v_m[neurons_number];
	init_array<float>(v_m, neurons_number, E_L);

	// neuron refractory time
	int nrn_ref_time[neurons_number];
	init_array<int>(nrn_ref_time, neurons_number, ms_to_step(3.0));

	// neuron refractory time timer
	int nrn_ref_time_timer[neurons_number];
	init_array<int>(nrn_ref_time_timer, neurons_number, -1);

	// neuron state -- has spike or not
	bool has_spike[neurons_number];
	init_array<bool>(has_spike, neurons_number, false);

	// prepare variable to keep synapses number per each neuron
	int synapses_number[neurons_number];
	// init_array<int> provided below in the metadata synapses loop

	// dimensionless quantity between 0 and 1 that is associated with potassium channel activation
	float n[neurons_number];
	init_array<float>(n, neurons_number, alpha_n / (alpha_n + beta_n));

	// dimensionless quantity between 0 and 1 that is associated with sodium channel activation
	float h[neurons_number];
	init_array<float>(h, neurons_number, alpha_h / (alpha_h + beta_h));

	// dimensionless quantity between 0 and 1 that is associated with sodium channel inactivation
	float m[neurons_number];
	init_array<float>(m, neurons_number, alpha_m / (alpha_m + beta_m));

	// excitatory synapse exponential conductance
	float g_exc[neurons_number];
	init_array<float>(g_exc, neurons_number, 0);

	// inhibitory synapse exponential conductance
	float g_inh[neurons_number];
	init_array<float>(g_inh, neurons_number, 0);




	// init connectomes

	/// connections which are inhibited by C=1. REMOVED AS [54:]
	// D1 -> G2
	connect_fixed_outdegree(D1_3, G2_1, 0.5, 13);
	connect_fixed_outdegree(D1_3, inh_group5, sim_time_in_steps, 0);	 // FixME FAKE connectome
	// D2 -> D3
	connect_fixed_outdegree(D2_3, D3_1, 0.5, 12.5);
	connect_fixed_outdegree(D2_3, D3_4, 0.5, 12.5);
	// D4 -> D5
	connect_fixed_outdegree(D4_3, D5_1, 1, 10);
	connect_fixed_outdegree(D4_3, D5_4, 1, 10);
	// G3 -> G4
	connect_fixed_outdegree(G3_1, G4_1, 1.0, 65);
	connect_fixed_outdegree(G3_1, inh_group5, sim_time_in_steps, 0);	 // FixME FAKE connectome
	connect_fixed_outdegree(G3_2, G4_1, 1.0, 65);
	connect_fixed_outdegree(G3_2, inh_group5, sim_time_in_steps, 0);	 // FixME FAKE connectome
	/// end

	init_extensor_flexor();
	init_ref_arc();

	int **gpu_synapses_post_nrn_id, **synapses_post_nrn_id = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay, **synapses_delay = (int **)malloc(datasize<int* >(neurons_number));
	int **gpu_synapses_delay_timer, **synapses_delay_timer = (int **)malloc(datasize<int* >(neurons_number));
	float **gpu_synapses_weight, **synapses_weight = (float **)malloc(datasize<float* >(neurons_number));

	// fill arrays of synapses
	for(int neuron_id = 0; neuron_id < neurons_number; neuron_id++) {
		int syn_count = static_cast<int>(metadatas.at(neuron_id).size());
		int tmp_synapses_post_nrn_id[syn_count];
		int tmp_synapses_delay[syn_count];
		int tmp_synapses_delay_timer[syn_count];
		float tmp_synapses_weight[syn_count];

		int syn_id = 0;
		for(SynapseMetadata metadata : metadatas.at(neuron_id)) {
			tmp_synapses_post_nrn_id[syn_id] = metadata.post_id;
			tmp_synapses_delay[syn_id] = metadata.synapse_delay;
			tmp_synapses_delay_timer[syn_id] = -1;
			tmp_synapses_weight[syn_id] = metadata.synapse_weight;
			syn_id++;
		}

		synapses_number[neuron_id] = syn_count;

		hipMalloc((void**)&synapses_post_nrn_id[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_delay_timer[neuron_id], datasize<int>(syn_count));
		hipMalloc((void**)&synapses_weight[neuron_id], datasize<float>(syn_count));

		hipMemcpy(synapses_post_nrn_id[neuron_id], &tmp_synapses_post_nrn_id, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay[neuron_id], &tmp_synapses_delay, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_delay_timer[neuron_id], &tmp_synapses_delay_timer, datasize<int>(syn_count), hipMemcpyHostToDevice);
		hipMemcpy(synapses_weight[neuron_id], &tmp_synapses_weight, datasize<float>(syn_count), hipMemcpyHostToDevice);
	}

	hipMalloc((void ***)&gpu_synapses_post_nrn_id, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_post_nrn_id, synapses_post_nrn_id, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay, synapses_delay, neurons_number);

	hipMalloc((void ***)&gpu_synapses_delay_timer, datasize<int *>(neurons_number));
	memcpyHtD<int *>(gpu_synapses_delay_timer, synapses_delay_timer, neurons_number);

	hipMalloc((void ***)&gpu_synapses_weight, datasize<float *>(neurons_number));
	memcpyHtD<float *>(gpu_synapses_weight, synapses_weight, neurons_number);

	hipMalloc(&gpu_v_m, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_v_m, v_m, neurons_number);

	hipMalloc(&gpu_has_spike, datasize<bool>(neurons_number));
	memcpyHtD<bool>(gpu_has_spike, has_spike, neurons_number);

	hipMalloc(&gpu_nrn_ref_time, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_time, nrn_ref_time, neurons_number);

	hipMalloc(&gpu_nrn_ref_time_timer, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_nrn_ref_time_timer, nrn_ref_time_timer, neurons_number);

	hipMalloc(&gpu_g_exc, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_g_exc, g_exc, neurons_number);

	hipMalloc(&gpu_g_inh, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_g_exc, g_inh, neurons_number);

	hipMalloc(&gpu_h, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_h, h, neurons_number);

	hipMalloc(&gpu_m, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_m, m, neurons_number);

	hipMalloc(&gpu_n, datasize<float>(neurons_number));
	memcpyHtD<float>(gpu_n, n, neurons_number);

	hipMalloc(&gpu_synapses_number, datasize<int>(neurons_number));
	memcpyHtD<int>(gpu_synapses_number, synapses_number, neurons_number);


	int threads_per_block = 1024;
	int num_blocks = neurons_number / threads_per_block + 1;

	printf("Size of network: %i \n", neurons_number);
	printf("Start GPU with %d threads x %d blocks (Total: %d threads) \n",
	       threads_per_block, num_blocks, threads_per_block * num_blocks);


	int activated_C_ = 0;
	int shift_time_by_step = 0;
	int master_local_iter = 0;

	// the main simulation loop
	for (int sim_iter = 0; sim_iter < sim_time_in_steps; sim_iter++) {
		// if flexor C0 activated, find the end of it and change to C1
		if (activated_C_ == 0) {
			if (master_local_iter != 0 && master_local_iter % steps_activation_C0 == 0) {
				activated_C_ = 1;
				master_local_iter = 0;
				// add const 125 ms
				shift_time_by_step += steps_activation_C0;
			}
			// if extensor C1 activated, find the end of it and change to C0
		} else {
			if (master_local_iter != 0 && master_local_iter % steps_activation_C1 == 0) {
				activated_C_ = 0;
				master_local_iter = 0;
				// add layers * 25 to the shift
				shift_time_by_step += steps_activation_C1;
			}
		}

		// printf("step %d [local %d] (%.2f ms) with C%d \n", sim_iter, master_local_iter, sim_iter * SIM_STEP, activated_C_);
		master_local_iter++;

		// invoke GPU
		GPU_simulation<<<num_blocks, threads_per_block>>>(
				gpu_v_m,
				gpu_h,
				gpu_m,
				gpu_n,
				gpu_g_exc,
				gpu_g_inh,
				gpu_has_spike,
				gpu_nrn_ref_time,
				gpu_nrn_ref_time_timer,
				gpu_synapses_number,
				gpu_synapses_post_nrn_id,
				gpu_synapses_delay,
				gpu_synapses_delay_timer,
				gpu_synapses_weight,
				sim_iter,
				activated_C_,
				shift_time_by_step);

		memcpyDtH<float>(v_m, gpu_v_m, neurons_number);
		memcpyDtH<float>(g_exc, gpu_g_exc, neurons_number);
		memcpyDtH<float>(g_inh, gpu_g_inh, neurons_number);
		memcpyDtH<bool>(has_spike, gpu_has_spike, neurons_number);

		// do some jobs with data

	} // end of the simulation iteration loop

	// tell the CPU to halt further processing until the CUDA kernel has finished doing its business
	hipDeviceSynchronize();
	hipDeviceReset();

//	save_result(test_index, voltage_recording, current_recording, spike_recording, neurons_number, full_save);

}

int main(int argc, char* argv[]) {
	simulate(std::atoi(argv[1]), std::atoi(argv[2]));

	return 0;
}